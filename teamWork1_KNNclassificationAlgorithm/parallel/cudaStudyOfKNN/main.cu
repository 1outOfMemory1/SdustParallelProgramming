
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;


//这个程序是用来测试 一个block中含有100个一维排布的线程 用于计算 一个二维矩阵(只不过存储是一维形式的) 减去一个相同列数的一维数组
/* 例子
10 * 10 的矩阵 aa
4  4  4  4  4  4  4  4  4
4  4  4  4  4  4  4  4  4
4  4  4  4  4  4  4  4  4
4  4  4  4  4  4  4  4  4
4  4  4  4  4  4  4  4  4
4  4  4  4  4  4  4  4  4
4  4  4  4  4  4  4  4  4
4  4  4  4  4  4  4  4  4
4  4  4  4  4  4  4  4  4


1 * 10 的矩阵 bb
1  1  1  1  1  1  1  1  1


10 * 10 的矩阵 result
3  3  3  3  3  3  3  3  3
3  3  3  3  3  3  3  3  3
3  3  3  3  3  3  3  3  3
3  3  3  3  3  3  3  3  3
3  3  3  3  3  3  3  3  3
3  3  3  3  3  3  3  3  3
3  3  3  3  3  3  3  3  3
3  3  3  3  3  3  3  3  3
3  3  3  3  3  3  3  3  3
 */

__global__ void sub(double *aa,double *bb,double *result){
    int xx = threadIdx.x;
//    int row = xx / 10;  row没有什么用
    int col = xx % 10;  //这里为了方便演示 所以直接模10了 可以将值直接传入其中
    result[xx] = aa[xx] - bb[col]; //这个是核心语句 定位到每个thread的xx参数范围是1-100 只有col是1-10
}

int main() {
    double *aa = new double[100];
    double *bb = new double[10];
    double *result = new double[100];
    for(int i=0;i<100;i++){
//        if(i%10 == 0 && i!=0)
//            cout<<endl;
        aa[i] = 4;
//        cout<<aa[i]<<" ";
    }
//    aa[23] = 13; //用来验证矩阵是否正确
    cout<<endl;
    for(int j=0;j<10;j++){
        bb[j] = 1;
    }
    double *cudaAA;
    double *cudaBB;
    double *cudaResult;
    hipMalloc((void**)&cudaAA,sizeof(double) *100);
    hipMalloc((void**)&cudaBB,sizeof(double) * 10);
    hipMalloc((void**)&cudaResult,sizeof(double) * 100);

    hipMemcpy(cudaAA,aa,sizeof(double) * 100,hipMemcpyHostToDevice);
    hipMemcpy(cudaBB,bb,sizeof(double) * 10,hipMemcpyHostToDevice);


    sub<<<1,100>>>(cudaAA,cudaBB,cudaResult);
    hipMemcpy(result,cudaResult,sizeof(double) * 100,hipMemcpyDeviceToHost);
    for(int i=0;i<100;i++){
        if(i%10 == 0 && i!=0)
            cout<<endl;
        cout<<result[i]<<" ";
    }
}
