#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include <ctime>
#include <cstdlib>
#include "yhncsv.h"
#include "common.h"
#include <map>

using namespace std;

// 全局变量
double trainDataProportion = (float )2/3;  //用于规定训练集占总数据的比例
int dataSize = 0;
int trainDataSize = 0; //用于记录训练集的大小
int testDataSize = 0; //用于记录测试集的大小
int columnSize = 0;  //用于记录数据列的数量
int threadSize = 2; //一个块中 线程数是32 * 32  =1024 最大值
int k=10; //  用来设置取前 k 个距离最近的数据
double allKernelFunctionCostTime = 0;
double allCostTime = 0;
string fileName = "wineQuality.csv";  //定义文件名字



//一次性算整个数组   测试集中的一行 都被训练集中的每一行先做减法然后平方
__global__ void MatrixSubAndSquare(double *trainSet,  //传入二维数组 每一个都可以
                                   double *oneRowOftestSet, //需要计算距离测试集的某一行
                                   double *afterSubAndSquareResultArray, //经过减法计算和平方计算后的中间数据
                                   int columnSize
                                   ){
    //设想的grid 分布  rowSize/ThreadSize columnSize/ThreadSize   ThreadSize=32 ThreadSize=32
    // rowSize/ThreadSize * ThreadSize = rowSize   columnSize/ThreadSize = columnSize
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int col = blockDim.y * blockIdx.y + threadIdx.y;
    double value = trainSet[row * columnSize + col]  - oneRowOftestSet[col]; //把平方后的值放在新数组中
    afterSubAndSquareResultArray[row * columnSize + col] = value * value;
}

__global__ void sumMatrix(double *aa,double *distance,int columnSize){ //计算每行的和 然后开方
    int x = blockIdx.x *blockDim.x + threadIdx.x;
    double value = 0;
    for(int i=0;i<columnSize;i++){
        value += aa[x * columnSize + i];
    }
    distance[x] = sqrt(value);
}


//这个knn函数的一次运行 算出的是  一行测试数据 距离  所有训练集所有行的距离 然后根据最近的k个数据来预测值
bool knn(vector<double> * testPiece, int position ,vector<vector<double>> *doubleDataVector,vector<string> * resultVector,set<string>* resultSet){
    //1. 初始化一些变量
    //1.1 基本变量的初始化
    double maxWeight = -1;  // 用来存储最大权重
    string maxWeightStr = "";  // 用来存储最大权重的字符串  也就是预测值
    bool flag = false;   // 返回给主函数 用于判断预测是否正确
    double sum = 0;  //这个数据用于之后计算权值的时候用 谁离得最近 权值越高
    auto *doubleArrayA = new double[trainDataSize * columnSize]; // 用于在传输数据的时候用数组临时存储训练集数组 vector行不通 把值逐个赋值给数组 vector不行
    auto *doubleArrayB = new double[columnSize]; //是用来存储测试集的一行数据的 和上方一样
    //    double * doubleArrayResult = new double[trainDataSize*columnSize];  //用于存储中间数据 经过相减平方后的数据 调试时可以输出
    //1.2 映射指针的创建 定义指针 用来映射显存中的数据
    double *cudaDoubleArray; //整个数据二维数组 显存中的数据
    double *cudaTestArrayPiece; //一行测试集数据 显存中的数据
    double *cudaAfterSubAndSquareDoubleArrayResult; //中间数据  显存中的数据
    //1.3 核函数规模的定义 第一个是矩阵减法 和 平方的核函数
    dim3 firstBlocksPerGrid(trainDataSize/threadSize,columnSize/threadSize);
    dim3 firstThreadsPerBlock(threadSize,threadSize);
    dim3 secondBlocksPerGrid(trainDataSize/threadSize);
    dim3 secondThreadsPerBlock(threadSize);
    //2. 申请空间
    hipMalloc((void**)&cudaDoubleArray,sizeof(double) * trainDataSize * columnSize ); //申请显存中二维数组的空间 用于存放训练集数据
    hipMalloc((void**)&cudaTestArrayPiece,sizeof(double) * columnSize);  //申请测试数据一维数组的空间 二维训练集的每一行都减去一维测试集的对应位的数据 然后平方
    hipMalloc((void**)&cudaAfterSubAndSquareDoubleArrayResult,sizeof(double) * trainDataSize * columnSize ); //申请中间结果的显存空间 规模和训练集一样
    //3.拷贝数据进入显存
    //3.1拷贝训练集显存
    //目前是没有什么好办法 只能挨个进行赋值 应该也不慢 但是肯定比那些直接进行内存整块拷贝的慢
    //3.1.1 先把数据弄到一个double数组中去
    for(int i=0;i<trainDataSize;i++){
        for(int j =0;j<columnSize;j++){
            doubleArrayA[i* columnSize +j] = doubleDataVector->at(i).at(j);
        }
    }
    //3.1.2 执行cuda显存拷贝函数
    hipMemcpy(cudaDoubleArray,doubleArrayA,sizeof(double)  * columnSize * trainDataSize ,hipMemcpyHostToDevice); //将训练集的数据拷入到显存中
    //3.2 拷贝测试集数据 只有一行 所以可以用copy函数
    //3.2.1 从vector<double> 转换为 double 数组
    copy(testPiece->begin(),testPiece->end(),doubleArrayB);  //分别表示 要复制的vector的头, 要复制的vector的尾 , 目标数组
    //3.2.2 执行cuda显存拷贝函数
    hipMemcpy(cudaTestArrayPiece,doubleArrayB ,sizeof(double) * columnSize ,hipMemcpyHostToDevice); //将test的数据传入
    //4. 执行第一个核函数
    hipEvent_t start1,stop1;
    float elapsedTime1 = 0;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventRecord(start1,0);
    MatrixSubAndSquare<<<firstBlocksPerGrid,firstThreadsPerBlock>>>(cudaDoubleArray,cudaTestArrayPiece,cudaAfterSubAndSquareDoubleArrayResult,columnSize);
    hipEventRecord(stop1,0);
    hipEventSynchronize(stop1);
    hipEventElapsedTime(&elapsedTime1,start1,stop1);
//    cout<<"first kennel function cost time:"<<elapsedTime1<<endl;

    //将结果拷贝回来  这一步是中间步骤 调试的时候排错用
//    hipMemcpy(doubleArrayResult,cudaAfterSubAndSquareDoubleArrayResult,trainDataSize * columnSize *sizeof(double)  ,hipMemcpyDeviceToHost); //将训练集的数据拷入到显存中)
//    //打印计算的中间结果 中间步骤 调试使用
//    for(int i=0;i<trainDataSize;i++){
//        for(int j=0;j<columnSize;j++){
//            cout<< doubleArrayResult[i*columnSize + j]<<"    ";
//        }
//        cout<<endl;
//    }
    //5. 释放一部分显存和内存  注意没有释放 cudaAfterSubAndSquareDoubleArrayResult 因为中间结果还需要使用
    //5.1 释放显存
    hipFree(cudaDoubleArray); //释放 二维数组(排布为一维) 训练集数据
    hipFree(cudaTestArrayPiece); //释放 一维数组 测试集的一行数据
    //5.2 释放内存
    free(doubleArrayA);
    free(doubleArrayB);
    //6 为执行第二个核函数准备空间(内存和显存)
    auto *distanceArray = new double[trainDataSize]; //申请内存空间 用来存放距离数组
    double *cudaDistanceArray; //申请空间 映射显存空间 用来存放距离数组
    hipMalloc((void**)&cudaDistanceArray,sizeof(double) * trainDataSize ); //申请存放距离显存空间
    //7 执行第二个核函数
    hipEvent_t start2,stop2;
    float elapsedTime2 = 0;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipEventRecord(start2,0);
    sumMatrix<<<secondBlocksPerGrid,secondThreadsPerBlock>>>(cudaAfterSubAndSquareDoubleArrayResult,cudaDistanceArray,columnSize);
    hipEventRecord(stop2,0);
    hipEventSynchronize(stop2);
    hipEventElapsedTime(&elapsedTime2,start2,stop2);
//    cout<<"second kennel function cost time:"<<elapsedTime2<<endl;
    allKernelFunctionCostTime += elapsedTime1+elapsedTime2;
    //8 将最后的距离数组拷贝回内存 以便后边使用
    hipMemcpy(distanceArray,cudaDistanceArray,sizeof(double) *trainDataSize ,hipMemcpyDeviceToHost);
    //9 释放掉所有显存 因为以后用不到了
    hipFree(cudaAfterSubAndSquareDoubleArrayResult);  //释放掉中间数据数组
    hipFree(cudaDistanceArray);  //释放掉距离数组
//    for(int i=0;i<trainDataSize ;i++ ){  //打印距离数据
//        cout<<distanceArray[i]<<endl;
//    }
    //10 最后的数据处理
    //10.1 初始化一些stl 以后会用到
    auto *realityAndDistanceMap = new multimap<double,string>;  //距离作为key 真实值为value 这样做的好处是自动排序 需要采用multimap 虽然距离一般不能一样 但是就怕巧了
    auto *weightMap = new map<string,double>;  //权重map   后边double数据可以作为依据 key不可能重复 所以放心用 map
    set<string>::iterator setItr ;  //用来遍历所有结果(resultSet) 这个set里存放了结果集的所有可能 比如判断是否得病的数据集 只有得病或者不得病两种 已经是排好序的
    map<double,string>::iterator mapIter; //用于遍历
    //10.2 计算好距离之后需要把数据和真实值对应起来  之后用于统计权值的时候回用到 realityAndDistanceMap multimap<double,string>
    for(int i=0;i<trainDataSize;i++){
        realityAndDistanceMap->insert(pair<double,string>(distanceArray[i] ,resultVector->at(i)));
    }
    free(distanceArray); //顺手把distanceArray释放掉 以后不会再用了
    //10.3.初始化权重map 把value都设成0  统计权重的目的是综合考虑k个最近的点的影响 约接近的点权重越高
    setItr = resultSet->begin();  //resultSet set<string> 的迭代器
    for(;setItr!=resultSet->end();setItr++){
        weightMap->insert(pair<string,double>(*setItr,0));
    }
    //10.4 计算前k个最近的点的总距离sum 用于算权重
    mapIter = realityAndDistanceMap->begin();
    for(int i=0;i<k;i++,mapIter++){ //计算sum值 计算出来sum值
        sum  += mapIter->first;
    }
    //10.5 分别计算前k个点的权值 根据其真实值 加到所有可能的值上 比如得病权重5.4 不得病权重为 8.4 所以可以判断大概率是不得病
    mapIter = realityAndDistanceMap->begin();
    for(int i=0;i<k;i++ ,mapIter++){  //前k个元素的权重算出来
        (*weightMap)[mapIter->second] += 1 - (mapIter->first / sum) ; //距离越近 权重越高  注意这里是 1- xxx
    }
    //10.6 找到最高的那个权重的值 比如是不得病 然后将它赋值给maxWeightStr
    for(pair<string,double> p :*weightMap ){
        if(p.second > maxWeight){
            maxWeight = p.second;
            maxWeightStr = p.first;
        }
    }
    //10.7 根据测试集的真实值和预测值对比是否一样  如果一样说明预测成功
    if(resultVector->at(position).compare(maxWeightStr)  == 0)
        flag = true;  //flag 后边会return回去
    else
        flag = false;
    //11 最终释放掉所有的内存(显存已经全部释放)
    free(realityAndDistanceMap); //释放真实值和距离map
    free(weightMap); //释放权重map
    return flag; //返回预测结果和真实值是否匹配
}


int main(int argc,char * argv[]) {
    clock_t allTimeBegin = clock();
    if(argc > 1){
        fileName = argv[1];
        cout<<"already input value， the csv file is:   "<<argv[1]<<endl<<endl;
    }else{
        cout<<"no input value!!!!  the default csv file is:"<<fileName<<endl;
    }

    //1. 初始化所有参数
    vector<vector<double>> *doubleDataVector = nullptr; //二维数组 用来存放训练集和测试集的所有数据
    vector<string> * resultVector = nullptr;  //结果集 里边全是字符串
    set<string> * realitySet = nullptr;  //无重复的所有结果集
    ifstream inputFile;   //定义文件输入流

    //2.处理i/o流 打开文件 然后调用yhnCsv类来读取csv文件的数据
    inputFile.open(fileName);  //打开文件
    Csv * csvReader = new Csv(&inputFile);  //把文件句柄传进去
    //3. 获取数据
    //3.1 数组或者集合的获取
    realitySet = csvReader->getResultSet();  //获取不重复的数据集
    resultVector = csvReader->getResultVector(); //拿到所有的结果
    doubleDataVector = csvReader->getDoubleData(); //把所有的训练集和测试集的数据拿到

    //3.2 获取全局变量
    columnSize = doubleDataVector->at(0).size();  //数据列的数量
    dataSize = doubleDataVector->size();  //记录总数据集的行数
    trainDataSize = trainDataProportion * dataSize;  //记录训练集的行数  比例乘以 总数据集的行数
    testDataSize = dataSize - trainDataSize; //记录测试集的行数
    //4. 同时对全部数据集和结果数据集进行随机 如果随机数不相同 那么就交换  这样能同时进行多个数组的交换 swap函数很好使
    srand((unsigned int)time(NULL));  //以时间为基准进行随机
    for (int i = 0; i < dataSize; ++i) {  //最多交换 总数据集大小 其实一般就行 不过无所谓了
        int n1 = (rand() % dataSize);//产生n以内的随机数  n是数组元素个数
        int n2 = (rand() % dataSize);
        if (n1 != n2) { //若两随机数不相等 则下标为这两随机数的数组进行交换
            swap(doubleDataVector->at(n1),doubleDataVector->at(n2));
            swap(resultVector->at(n1),resultVector->at(n2));
        }
    }
    //csvReader->printHeaderVector();  //打印头的所有字符串 不包括结果列的名字
    //csvReader->printResultInformation(); //打印result信息
    cout<<"The k value is："<<k<<",the all dataSet has "<<dataSize<<" pieces of data，"<<"the train Set has "<<trainDataSize<<",the test set has "<<testDataSize<<endl;
    int count = 0; //用来统计成功预测的数量
    for(int i=0;i<testDataSize;i++){
        bool flag =  knn(&doubleDataVector->at(trainDataSize + i), //测试集的一行
                         trainDataSize +i  , //测试集的位置
                         doubleDataVector, // 所有的数据
                         resultVector, //结果的数据集
                         realitySet); //结果的所有可能值 的 set(不重复)
//        bool flag =  knn(&doubleDataVector->at(69), //测试集的一行
//                         69 , //测试集的位置
//                         doubleDataVector, // 所有的数据
//                         resultVector, //结果的数据集
//                         realitySet); //结果的所有可能值 的 set(不重复)
        if(flag){
            count++;
        }
    }
    clock_t allTimeEnd = clock();
    cout<<"this time the accuracy of dataSet is "<<(float )count/testDataSize *100 <<"%"<<endl ;
    cout<<"all Kernel Function Cost Time: "<<allKernelFunctionCostTime<<" ms"<<endl;
    cout<<"total Cost Time:"<<(allTimeEnd - allTimeBegin)/CLOCKS_PER_SEC <<" s"<<endl;

    free(csvReader);
}
