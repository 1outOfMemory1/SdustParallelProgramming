#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <opencv2/opencv.hpp>

using namespace cv;
using namespace std;

// 图像的宽高
const int row = 700;
const int col = 1400;
// 25X25
const int x = 25;
const int y = 25;
// 二维数组
int ** myMalloc(int row, int col) {
    int** arr = new int*[row];
    for (int i = 0; i < row; i++) {
        arr[i] = new int[col];
    }
    return arr;
}
// Mat转换成二维数组
int ** mat2Array(Mat mat) {
    int** arr = myMalloc(mat.rows, mat.cols);
    for (int i = 0; i < mat.rows; i++) {
        for (int j = 0; j < mat.cols; j++) {
            arr[i][j] = mat.at<uchar>(i, j);
        }
    }
    return arr;
}
// 二维数组转换成Mat
Mat array2Mat(int arr[row][col], int row, int col) {
    Mat mat(row, col, CV_8UC1);
    unsigned char *pTmp = NULL;
    for (int i = 0; i < row; i++) {
        pTmp = mat.ptr(i);
        for (int j = 0; j < col; j++) {
            pTmp[j] = arr[i][j];
        }
    }
    return mat;
}
// 数乘
__global__ void numMul(int a[row][col], int num, int result[row][col]) {
    int colG = blockDim.x * blockIdx.x + threadIdx.x;
    int rowG = blockDim.y * blockIdx.y + threadIdx.y;
    result[colG][rowG] = a[colG][rowG] * num;
}

// 矩阵相加
__global__ void matAdd1(int a[row][col], int b[row][col], int result[row][col]) {
    int colG = blockDim.x * blockIdx.x + threadIdx.x;
    int rowG = blockDim.y * blockIdx.y + threadIdx.y;
    result[rowG][colG] = a[rowG][colG] + b[rowG][colG];
}
__global__ void matAdd2(int a[row][col], int num, int result[row][col]) {
    int colG = blockDim.x * blockIdx.x + threadIdx.x;
    int rowG = blockDim.y * blockIdx.y + threadIdx.y;
    result[rowG][colG] = a[rowG][colG] + num;
}
// 除
__global__ void matDiv(int a[row][col], int num, int result[row][col]) {
    int colG = blockDim.x * blockIdx.x + threadIdx.x;
    int rowG = blockDim.y * blockIdx.y + threadIdx.y;
    result[rowG][colG] = a[rowG][colG] / num;
}

int main() {
    Mat src = imread("in.jpg");
    // 通道分割
    vector<Mat> channels;
    split(src, channels);
    Mat B = channels[0];
    Mat G = channels[1];
    Mat R = channels[2];
    // mat2array
    int **bArray = mat2Array(B);
    int **gArray = mat2Array(G);
    int **rArray = mat2Array(R);
    // 分配存储空间
    int (*tmp1)[col];
    int (*tmp2)[col];
    int (*tmp3)[col];
    int (*result1)[col];
    int (*result2)[col];
    int (*result3)[col];
    int (*result4)[col];
    int (*result5)[col];
    int (*result6)[col];
    int (*result7)[col];
    hipMalloc((void**)&tmp1, sizeof(int)*row*col);
    hipMalloc((void**)&tmp2, sizeof(int)*row*col);
    hipMalloc((void**)&tmp3, sizeof(int)*row*col);
    hipMalloc((void**)&result1, sizeof(int)*row*col);
    hipMalloc((void**)&result2, sizeof(int)*row*col);
    hipMalloc((void**)&result3, sizeof(int)*row*col);
    hipMalloc((void**)&result4, sizeof(int)*row*col);
    hipMalloc((void**)&result5, sizeof(int)*row*col);
    hipMalloc((void**)&result6, sizeof(int)*row*col);
    hipMalloc((void**)&result7, sizeof(int)*row*col);
    int (*bArray_c)[col] = new int[row][col];
    int (*gArray_c)[col] = new int[row][col];
    int (*rArray_c)[col] = new int[row][col];
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            bArray_c[i][j] = bArray[i][j];
            gArray_c[i][j] = gArray[i][j];
            rArray_c[i][j] = rArray[i][j];
        }
    }
    hipMemcpy(tmp1, bArray_c, sizeof(int)*row*col, hipMemcpyHostToDevice);
    hipMemcpy(tmp2, gArray_c, sizeof(int)*row*col, hipMemcpyHostToDevice);
    hipMemcpy(tmp3, rArray_c, sizeof(int)*row*col, hipMemcpyHostToDevice);
    // =========
    dim3 threadsPerBlock(x, y);
    // =========
    dim3 blocksPerGrid(row / threadsPerBlock.x, col / threadsPerBlock.y);
    // ====
    hipEvent_t start, finish;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&finish);
    hipEventRecord(start, 0);

    numMul <<< blocksPerGrid, threadsPerBlock >>> (tmp1, 299, result1);
    numMul <<< blocksPerGrid, threadsPerBlock >>> (tmp2, 587, result2);
    numMul <<< blocksPerGrid, threadsPerBlock >>> (tmp3, 114, result3);
    matAdd1 <<< blocksPerGrid, threadsPerBlock >>> (result1, result2, result4);
    matAdd1 <<< blocksPerGrid, threadsPerBlock >>> (result3, result4, result5);
    matAdd2 <<< blocksPerGrid, threadsPerBlock >>> (result5, 500, result6);
    matDiv <<< blocksPerGrid, threadsPerBlock >>> (result6, 1000, result7);

    hipEventRecord(finish, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(finish);
    hipEventElapsedTime(&elapsedTime, start, finish);
    cout << elapsedTime << " ms" << endl;

    // ====
    int (*resultArray)[col] = new int[row][col];
    hipMemcpy(resultArray, result7, sizeof(int)*row*col, hipMemcpyDeviceToHost);
    Mat result = array2Mat(resultArray, row, col);
    // 保存图片
    imwrite("out.jpg", result);
    // 释放存储空间
    hipFree(tmp1);
    hipFree(tmp2);
    hipFree(tmp3);
    hipFree(result1);
    hipFree(result2);
    hipFree(result3);
    hipFree(result4);
    hipFree(result5);
    hipFree(result6);
    hipFree(result7);
    delete [] resultArray;
    return 0;
}