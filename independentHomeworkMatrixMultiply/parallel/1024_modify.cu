
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
const int n = 1024;
//当n大于1024时，需要对块的维度和网格的维度做一些调整
//每个块内有32*32个线程，网格中共有(n/32)*(n/32)个块
__global__ void kernel(int * d_a,int * d_b,int * d_result)
{
    //两维的d_result[][]矩阵存放结果，blockDim.y*blockIdx.y+threadIdx.y代表线程所处理的行坐标，
    //blockDim.x*blockIdx.x+threadIdx.x代表线程所处理的列坐标
    int r=blockDim.y*blockIdx.y+threadIdx.y;
    int c=blockDim.x*blockIdx.x+threadIdx.x;
    d_result[r*n+c] = 0;
    for(int i=0;i<n;i++)
        d_result[r*n+c]+=d_a[r*n+i]*d_b[i*n+c];
}
int main()
{
    //指向CPU端内存的指针
    int  *h_a = new int[n*n];
    int  *h_b = new int[n*n];
    int  *h_result = new int[n*n];

    //为两个矩阵赋初值
    for(int i=0;i<n*n;i++){
        h_a[i] = 1;
        h_b[i] = 1;
    }



    //指向GPU端内存的指针
    int * d_a , *d_b , *d_result ;

    //为GPU中的数据分配内存
    hipMalloc( (void**)&d_a,sizeof(int)*n*n  );
    hipMalloc( (void**)&d_b,sizeof(int)*n*n  );
    hipMalloc( (void**)&d_result,sizeof(int)*n*n  );

    //拷贝CPU中的数据到GPU
    hipMemcpy(d_a,h_a,sizeof(int)*n*n,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,sizeof(int)*n*n,hipMemcpyHostToDevice);

    //调用内核函数
    kernel<<<dim3(n/32,n/32),dim3(32,32)>>>(d_a,d_b,d_result);


    //把GPU中算出来的数据拷回CPU
    hipMemcpy(h_result,d_result,sizeof(int)*n*n,hipMemcpyDeviceToHost);

    //显示
    for(int i=0;i<n;i++)
        for(int j=0;j<n;j++)
        {
            cout<<h_result[i*n+j]<<"   ";
            if(j==n-1)
                cout<<'\n';
        }

}
