
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;


//����������������� һ��block�к���100��һά�Ų����߳� ���ڼ��� һ����ά����(ֻ�����洢��һά��ʽ��) ��ȥһ����ͬ������һά����
/* ����
10 * 10 �ľ��� aa
4  4  4  4  4  4  4  4  4
4  4  4  4  4  4  4  4  4
4  4  4  4  4  4  4  4  4
4  4  4  4  4  4  4  4  4
4  4  4  4  4  4  4  4  4
4  4  4  4  4  4  4  4  4
4  4  4  4  4  4  4  4  4
4  4  4  4  4  4  4  4  4
4  4  4  4  4  4  4  4  4


1 * 10 �ľ��� bb
1  1  1  1  1  1  1  1  1


10 * 10 �ľ��� result
3  3  3  3  3  3  3  3  3
3  3  3  3  3  3  3  3  3
3  3  3  3  3  3  3  3  3
3  3  3  3  3  3  3  3  3
3  3  3  3  3  3  3  3  3
3  3  3  3  3  3  3  3  3
3  3  3  3  3  3  3  3  3
3  3  3  3  3  3  3  3  3
3  3  3  3  3  3  3  3  3
 */

__global__ void sub(double *aa,double *bb,double *result){
    int xx = threadIdx.x;
//    int row = xx / 10;  rowû��ʲô��
    int col = xx % 10;  //����Ϊ�˷�����ʾ ����ֱ��ģ10�� ���Խ�ֱֵ�Ӵ�������
    result[xx] = aa[xx] - bb[col]; //����Ǻ������ ��λ��ÿ��thread��xx������Χ��1-100 ֻ��col��1-10
}

int main() {
    double *aa = new double[100];
    double *bb = new double[10];
    double *result = new double[100];
    for(int i=0;i<100;i++){
//        if(i%10 == 0 && i!=0)
//            cout<<endl;
        aa[i] = 4;
//        cout<<aa[i]<<" ";
    }
//    aa[23] = 13; //������֤�����Ƿ���ȷ
    cout<<endl;
    for(int j=0;j<10;j++){
        bb[j] = 1;
    }
    double *cudaAA;
    double *cudaBB;
    double *cudaResult;
    hipMalloc((void**)&cudaAA,sizeof(double) *100);
    hipMalloc((void**)&cudaBB,sizeof(double) * 10);
    hipMalloc((void**)&cudaResult,sizeof(double) * 100);

    hipMemcpy(cudaAA,aa,sizeof(double) * 100,hipMemcpyHostToDevice);
    hipMemcpy(cudaBB,bb,sizeof(double) * 10,hipMemcpyHostToDevice);


    sub<<<1,100>>>(cudaAA,cudaBB,cudaResult);
    hipMemcpy(result,cudaResult,sizeof(double) * 100,hipMemcpyDeviceToHost);
    for(int i=0;i<100;i++){
        if(i%10 == 0 && i!=0)
            cout<<endl;
        cout<<result[i]<<" ";
    }
}
