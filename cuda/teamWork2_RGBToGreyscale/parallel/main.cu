#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "opencv2/core.hpp"
#include "opencv2/highgui.hpp"
#include <iostream>
#include <string>
using namespace cv;
using namespace std;


// global variables
int pictureHeight,pictureWidth; // define how many pixels of picture in the row and column
string pictureUrl = "D:/in.jpg";
int threadSize = 32;
__global__ void numberMultiplyMatrix(unsigned char * uCharArrayB,  // Initial array
                                     unsigned char * uCharArrayG,  // Initial array
                                     unsigned char * uCharArrayR,  // Initial array
                                     unsigned char * uCharArrayGrey,  // Result array that after multiply a coefficient
                                     int picWidth){
    // B 0.114   G 0.587  R 0.299
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int col = blockDim.y * blockIdx.y + threadIdx.y;

    uCharArrayGrey[row * picWidth + col] = (15*uCharArrayB[row * picWidth + col] +
                                            75*uCharArrayG[row * picWidth + col] +
                                            38*uCharArrayR[row * picWidth + col] )>> 7;
}
int main(int argc,char * argv[])
{
    double seconds;
    string fileName = "";
    clock_t begin_time = clock();
    if(argc > 1){
        pictureUrl = argv[1];
        cout<<"already input value,file name is "<<pictureUrl<<endl;
    }else{
        cout<<"no input value found,default picture file path is "<<pictureUrl<<endl;
    }
    int i=pictureUrl.size()-1;
    for(;i>=0;i--){
        if(pictureUrl[i] == '/' || pictureUrl[i] == '\\' ){
            fileName = pictureUrl.substr(i+1,pictureUrl.size());
            break;
        }
    }
    if(i <= 0)
        fileName = pictureUrl;


    Mat rawPic = imread(pictureUrl);
    Mat greyPic(rawPic.rows, rawPic.cols, CV_8UC1, Scalar(0));
    if(rawPic.empty()){
        cout<<"input picture not found, please check your path"<<endl;
        exit(-1);
    }
    pictureHeight = rawPic.rows;
    pictureWidth = rawPic.cols;
    unsigned char *uCharArrayB = new unsigned char[pictureHeight *pictureWidth]; // host memory
    unsigned char *uCharArrayG = new unsigned char[pictureHeight *pictureWidth]; // host memory
    unsigned char *uCharArrayR = new unsigned char[pictureHeight *pictureWidth]; // host memory
    unsigned char *uCharArrayGrey = new unsigned char[pictureHeight *pictureWidth]; // host memory
    unsigned char *cudaUCharArrayB; // cuda memory
    unsigned char *cudaUCharArrayG; // cuda memory
    unsigned char *cudaUCharArrayR; // cuda memory
    unsigned char *cudaUCharArrayGrey; // cuda memory
    //assign 3 channels values to three unsigned char array
    for (int i = 0; i < pictureHeight; i++)
    {
        unsigned char *cp = rawPic.ptr<uchar>(i);
        for(int j = 0; j < pictureWidth; j++){
            uCharArrayB[i * pictureWidth + j] = cp[0];
            uCharArrayG[i * pictureWidth + j] = cp[1];
            uCharArrayR[i * pictureWidth + j] = cp[2];
            cp+=3;
        }
    }
    // apply for display memory
    hipMalloc((void**)&cudaUCharArrayB,sizeof(unsigned char) * pictureWidth * pictureHeight); //
    hipMalloc((void**)&cudaUCharArrayG,sizeof(unsigned char) * pictureWidth * pictureHeight);  //
    hipMalloc((void**)&cudaUCharArrayR,sizeof(unsigned char) * pictureWidth * pictureHeight); //
    hipMalloc((void**)&cudaUCharArrayGrey,sizeof(unsigned char) * pictureWidth * pictureHeight); //
    // copy data from host memory to display memory
    hipMemcpy(cudaUCharArrayB,uCharArrayB,pictureHeight * pictureWidth * sizeof(unsigned char),hipMemcpyHostToDevice);
    hipMemcpy(cudaUCharArrayG,uCharArrayG,pictureHeight * pictureWidth * sizeof(unsigned char),hipMemcpyHostToDevice);
    hipMemcpy(cudaUCharArrayR,uCharArrayR,pictureHeight * pictureWidth * sizeof(unsigned char),hipMemcpyHostToDevice);
    hipEvent_t start,stop;
    float elapsedTime = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    // execute cuda  kernel function
    numberMultiplyMatrix<<<dim3(pictureHeight/threadSize,
                                pictureWidth/threadSize),
                                dim3(threadSize,threadSize)>>>
                                (cudaUCharArrayB,cudaUCharArrayG,cudaUCharArrayR,cudaUCharArrayGrey,pictureWidth);
    hipEventRecord(stop,0);
    hipMemcpy(uCharArrayGrey,cudaUCharArrayGrey,pictureHeight * pictureWidth * sizeof(unsigned char),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);
    printf("cost time : %f ms $$$$ %f s \n",elapsedTime,elapsedTime/1000);
    // copy the result to opencv Mat type ,in order to show the picture
    for (int i = 0; i < pictureHeight; i++)
    {
        unsigned char *cp = greyPic.ptr<uchar>(i);
        for(int j = 0; j < pictureWidth; j++){
            cp[0] =  uCharArrayGrey[i*pictureWidth + j];
            cp++;
        }
    }
    // show pictures
//    imshow("init",rawPic);
//    imshow("grey",greyPic);
//    waitKey(0);
    // convert Mat type to picture (type jpg)
    fileName = "out_" + fileName;
    imwrite(fileName,greyPic);
    cout<<"generate Grayscale image success,the output picture file name is "<<fileName<<endl;
    clock_t end_time = clock();
    seconds = ((double)end_time - begin_time) / CLOCKS_PER_SEC;
    hipFree(cudaUCharArrayB);
    hipFree(cudaUCharArrayG);
    hipFree(cudaUCharArrayR);
    hipFree(cudaUCharArrayGrey);
    delete[] uCharArrayB;
    delete[] uCharArrayG;
    delete[] uCharArrayR;
    delete[] uCharArrayGrey;
    cout<<"cost total time "<<seconds<<" seconds"<<endl;
    return 0;
}