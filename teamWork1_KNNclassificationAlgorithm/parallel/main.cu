
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <sstream>
#include <algorithm>

using namespace std;
__global__ void distance(){

}

__global__ void knn(){

}


vector<string>* getStringVector(string str,char symbol){
    vector<string> *stringVector = new vector<string>;
    vector<int> * positionOfSymbolVector = new vector<int>;
    int positionOfSymbolVectorSize = 0;
    for(int i=0;i<str.length();i++){
        if(str[i] == symbol){
            positionOfSymbolVector->push_back(i);
        }
    }
    positionOfSymbolVectorSize = positionOfSymbolVector->size(); //��ȡvector��С
    //��ȡ����һ�� �ַ���
    string temp = str.substr(0,positionOfSymbolVector->at(0));
    stringVector->push_back(temp);
    //ȡ���м�Ĳ���
    for(int i=0;i<positionOfSymbolVectorSize -1;i++){
        temp = str.substr(positionOfSymbolVector->at(i)+1,positionOfSymbolVector->at(i+1) - positionOfSymbolVector->at(i) -1);
        stringVector->push_back(temp);
    }
    //ȡ�����һ���ַ���
    temp = str.substr(positionOfSymbolVector->at(positionOfSymbolVectorSize -1) + 1,
                      str.size()- positionOfSymbolVector->at(positionOfSymbolVectorSize -1));
    stringVector->push_back(temp);
//    cout<<"size of stringVector:"<<stringVectorSize<<endl;
    free(positionOfSymbolVector); //�ͷ��ڴ��ֹ�ڴ�й©
    return stringVector;
}

vector<double>* getDoubleVector(string str,char symbol){
    vector<double> *doubleVector = new vector<double>;
    vector<int> * positionOfSymbolVector = new vector<int>;
    int positionOfSymbolVectorSize = 0;
    int flag = 0;
    for(int i=0;i<str.length();i++){
        if(str[i] == symbol){
            positionOfSymbolVector->push_back(i);
        }
    }
    positionOfSymbolVectorSize = positionOfSymbolVector->size(); //��ȡvector��С
    //��ȡ����һ�� �ַ���
    string temp = str.substr(0,positionOfSymbolVector->at(0));
    doubleVector->push_back(atof(temp.c_str()));
    //ȡ���м�Ĳ���
    for(int i=0;i<positionOfSymbolVectorSize -1;i++){
        temp = str.substr(positionOfSymbolVector->at(i)+1,positionOfSymbolVector->at(i+1) - positionOfSymbolVector->at(i) -1);
        doubleVector->push_back(atof(temp.c_str()));
    }
    //ȡ�����һ���ַ���
    temp = str.substr(positionOfSymbolVector->at(positionOfSymbolVectorSize -1) + 1,
                      str.size()- positionOfSymbolVector->at(positionOfSymbolVectorSize -1));
    doubleVector->push_back(atof(temp.c_str()));
//    cout<<"size of stringVector:"<<stringVectorSize<<endl;
    free(positionOfSymbolVector); //�ͷ��ڴ��ֹ�ڴ�й©
    return doubleVector;
}

//��Ϊvector�������洢���ݵ� ����ֱ�ӽ����ڴ濽������
//memcpy(doubleArray,&doubleVector[0],doubleVectorSize * sizeof(double));


int main() {
    const char symbol = ',';
    vector<string> *rowVector = new vector<string>;
    ifstream inputFile;
    string fileName = "../glass.csv";
    inputFile.open(fileName);
    string row;
    if(!inputFile.is_open()){
        cout<<"���ļ�ʧ�� open file failure"<<endl;
        exit(-1);
    }else{
        while (getline(inputFile,row)){
            // cout<<row<<endl;
            // �õ��������������� RI,Na,Mg,Al,Si,K,Ca,Ba,Fe,Type
            //1.52101,13.64,4.49,1.1,71.78,0.06,8.75,0,0,1
            rowVector->push_back(row);
        }
    }
    //��ȡ��һ��header����Ϣ
    vector<string> * header = getStringVector(rowVector->at(0),symbol);
    for(string temp : *header){
        cout<<temp<<" ";
    }
    cout<<endl;
    vector<double> * doubleVector = getDoubleVector(rowVector->at(1),symbol);
    for(double temp : *doubleVector){
        cout<<temp<<" ";
    }


}
