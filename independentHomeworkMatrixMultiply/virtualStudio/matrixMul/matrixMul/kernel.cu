#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
int arrayScale = 2; //���þ����ģ ȫ�ֱ���  
int arrayScale_square = arrayScale * arrayScale;  //��������ģ��ƽ�� ֮��ĳ�����õ�
#define size 1  //���ֵ�Ǹ���һ�������ж��ٸ��̵߳�  �����õ��Ƕ�ά��thread�Ų� 10 *10 Ϊ100 < 1024 ��Ϊ��ʦ�������ݶ���10�ı��� ��������10�ܺ���
using namespace std;


__global__ void MatMul(int* M, int* N, int* P, int scale)  //�����ĺ��ĺ��� �����Դ��е�A B���� result���� �� �����ģ
{
    //��ʵ���г�����Ƶ�Ŀ�ľ���ͬʱ���� �����������ģ��10 * 10  ��ô��Ҫ�õ�100���̼߳���result�����ÿһ��ֵ
    //���Բ��г���ĺ����Ƕ�λ����100���߳� (��ά��ά����ά����һά) Ȼ��Ѽ�������Ϣ���뵽�Դ��� 
    int Col = blockIdx.x * blockDim.x + threadIdx.x; // cloumn �����ǽ�4ά ��ά�� 2ά  ȥ��block�ı߿�������� ��һ���Ƕ�λ����һ��
    int Row = blockIdx.y * blockDim.y + threadIdx.y; // row   ��һ���Ƕ�λ����һ���� 
    float elem1 = 0.0, elem2 = 0.0, value = 0.0;
    for (int i = 0; i < scale; i++)
    {
        elem1 = M[Row * scale + i];//ȡM�����һ�� 
        elem2 = N[i * scale + Col];//ȡN�����һ��
        value += elem1 * elem2;//���
    }
    P[ Row * scale + Col] = value;
}


int main(int argc,char * argv[])
{

    if(argc > 1){
        int hhh = atoi(argv[1]); //��ȡִ��ʱ���� ������ת��Ϊintֵ ���ֵ��������С size * size ��С�������������
        arrayScale = hhh;
        arrayScale_square = arrayScale * arrayScale;
        cout<<"����������� �����ģΪ"<<arrayScale<<" * "<<arrayScale<<endl;
    }else{
        cout<<"δ������������� Ĭ�Ͼ����ģΪ"<<arrayScale<<" * "<<arrayScale<<endl;
    }
    
    int *intArrayA = new int[arrayScale_square];
    int *intArrayB = new int[arrayScale_square];
    int *intArrayResult = new int[arrayScale_square];

    int *gpuMappingIntArrayA,*gpuMappingIntArrayB,*gpuMappingIntArrayResult;

    dim3 blocksPerGrid(arrayScale/size,arrayScale/size);
    dim3 threadsPerBock(size,size);

    hipEvent_t start,stop;
    float elapsedTime = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //�豸���ڴ����

    hipMalloc((void**)&gpuMappingIntArrayA,arrayScale_square * sizeof(int));
    hipMalloc((void**)&gpuMappingIntArrayB,arrayScale_square * sizeof(int));
    hipMalloc((void**)&gpuMappingIntArrayResult,arrayScale_square * sizeof(int));


    //��ʼ��
    for(int i = 0;i < arrayScale;i++)
    {
        for(int j = 0;j < arrayScale;j++)
        {
            intArrayA[i*arrayScale + j] = 1;
            intArrayB[i*arrayScale + j] = 2;
        }
    }
    intarraya[2] = 10;
    intarraya[3] = 3;
    intarrayb[3] = 1;


    //���ݿ������������豸
    hipMemcpy(gpuMappingIntArrayA,intArrayA,arrayScale_square * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(gpuMappingIntArrayB,intArrayB,arrayScale_square * sizeof(int),hipMemcpyHostToDevice);

    hipEventRecord(start,0);
    MatMul<<<blocksPerGrid,threadsPerBock>>>(gpuMappingIntArrayA,gpuMappingIntArrayB,gpuMappingIntArrayResult,arrayScale);//���ú˺���
    hipDeviceSynchronize();
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);
    hipMemcpy(intArrayResult,gpuMappingIntArrayResult,arrayScale_square * sizeof(int),hipMemcpyDeviceToHost);



    printf("cost time : %f ms $$$$ %f s \n",elapsedTime,elapsedTime/1000);
   for(int i=0;i<arrayScale;i++){
       for(int j=0;j<arrayScale;j++){
            printf("%d ",intArrayResult[i*arrayScale + j]);
        }
       printf("\n");
    }


    //�ͷ��豸�ڴ�
    hipFree(gpuMappingIntArrayA);
    hipFree(gpuMappingIntArrayB);
    hipFree(gpuMappingIntArrayResult);
    free(intArrayA);
    free(intArrayB);
    free(intArrayResult);
    return 0;
}
