#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include <ctime>
#include <cstdlib>
#include "yhncsv.h"
#include "common.h"
#include <map>

using namespace std;

// ȫ�ֱ���
double trainDataProportion = (float )2/3;  //���ڹ涨ѵ����ռ�����ݵı���
int dataSize = 0;
int trainDataSize = 0; //���ڼ�¼ѵ�����Ĵ�С
int testDataSize = 0; //���ڼ�¼���Լ��Ĵ�С
int columnSize = 0;  //���ڼ�¼�����е�����
int threadSize = 2; //һ������ �߳�����32 * 32  =1024 ���ֵ
int k=14; //  ��������ȡǰ k ���������������
double allKernelFunctionCostTime = 0;
double allCostTime = 0;
string fileName = "F://fashion-mnist_train.csv";  //�����ļ�����



//һ��������������   ���Լ��е�һ�� ����ѵ�����е�ÿһ����������Ȼ��ƽ��
__global__ void MatrixSubAndSquare(double *trainSet,  //�����ά���� ÿһ��������
                                   double *oneRowOftestSet, //��Ҫ���������Լ���ĳһ��
                                   double *afterSubAndSquareResultArray, //�������������ƽ���������м�����
                                   int columnSize
                                   ){
    //�����grid �ֲ�  rowSize/ThreadSize columnSize/ThreadSize   ThreadSize=32 ThreadSize=32
    // rowSize/ThreadSize * ThreadSize = rowSize   columnSize/ThreadSize = columnSize
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int col = blockDim.y * blockIdx.y + threadIdx.y;
    double value = trainSet[row * columnSize + col]  - oneRowOftestSet[col]; //��ƽ�����ֵ������������
    afterSubAndSquareResultArray[row * columnSize + col] = value * value;
}

__global__ void sumMatrix(double *aa,double *distance,int columnSize){ //����ÿ�еĺ� Ȼ�󿪷�
    int x = blockIdx.x *blockDim.x + threadIdx.x;
    double value = 0;
    for(int i=0;i<columnSize;i++){
        value += aa[x * columnSize + i];
    }
    distance[x] = sqrt(value);
}


//���knn������һ������ �������  һ�в������� ����  ����ѵ���������еľ��� Ȼ����������k��������Ԥ��ֵ
bool knn(vector<double> * testPiece, int position ,vector<vector<double>> *doubleDataVector,vector<string> * resultVector,set<string>* resultSet){
    //1. ��ʼ��һЩ����
    //1.1 ���������ĳ�ʼ��
    double maxWeight = -1;  // �����洢���Ȩ��
    string maxWeightStr = "";  // �����洢���Ȩ�ص��ַ���  Ҳ����Ԥ��ֵ
    bool flag = false;   // ���ظ������� �����ж�Ԥ���Ƿ���ȷ
    double sum = 0;  //�����������֮�����Ȩֵ��ʱ���� ˭������ ȨֵԽ��
    auto *doubleArrayA = new double[trainDataSize * columnSize]; // �����ڴ������ݵ�ʱ����������ʱ�洢ѵ�������� vector�в�ͨ ��ֵ�����ֵ������ vector����
    auto *doubleArrayB = new double[columnSize]; //�������洢���Լ���һ�����ݵ� ���Ϸ�һ��
    //    double * doubleArrayResult = new double[trainDataSize*columnSize];  //���ڴ洢�м����� �������ƽ��������� ����ʱ�������
    //1.2 ӳ��ָ��Ĵ��� ����ָ�� ����ӳ���Դ��е�����
    double *cudaDoubleArray; //�������ݶ�ά���� �Դ��е�����
    double *cudaTestArrayPiece; //һ�в��Լ����� �Դ��е�����
    double *cudaAfterSubAndSquareDoubleArrayResult; //�м�����  �Դ��е�����
    //1.3 �˺�����ģ�Ķ��� ��һ���Ǿ������ �� ƽ���ĺ˺���
    dim3 firstBlocksPerGrid(trainDataSize/threadSize,columnSize/threadSize);
    dim3 firstThreadsPerBlock(threadSize,threadSize);
    dim3 secondBlocksPerGrid(trainDataSize/threadSize);
    dim3 secondThreadsPerBlock(threadSize);
    //2. ����ռ�
    hipMalloc((void**)&cudaDoubleArray,sizeof(double) * trainDataSize * columnSize ); //�����Դ��ж�ά����Ŀռ� ���ڴ��ѵ��������
    hipMalloc((void**)&cudaTestArrayPiece,sizeof(double) * columnSize);  //�����������һά����Ŀռ� ��άѵ������ÿһ�ж���ȥһά���Լ��Ķ�Ӧλ������ Ȼ��ƽ��
    hipMalloc((void**)&cudaAfterSubAndSquareDoubleArrayResult,sizeof(double) * trainDataSize * columnSize ); //�����м������Դ�ռ� ��ģ��ѵ����һ��
    //3.�������ݽ����Դ�
    //3.1����ѵ�����Դ�
    //Ŀǰ��û��ʲô�ð취 ֻ�ܰ������и�ֵ Ӧ��Ҳ���� ���ǿ϶�����Щֱ�ӽ����ڴ����鿽������
    //3.1.1 �Ȱ�����Ū��һ��double������ȥ
    for(int i=0;i<trainDataSize;i++){
        for(int j =0;j<columnSize;j++){
            doubleArrayA[i* columnSize +j] = doubleDataVector->at(i).at(j);
        }
    }
    //3.1.2 ִ��cuda�Դ濽������
    hipMemcpy(cudaDoubleArray,doubleArrayA,sizeof(double)  * columnSize * trainDataSize ,hipMemcpyHostToDevice); //��ѵ���������ݿ��뵽�Դ���
    //3.2 �������Լ����� ֻ��һ�� ���Կ�����copy����
    //3.2.1 ��vector<double> ת��Ϊ double ����
    copy(testPiece->begin(),testPiece->end(),doubleArrayB);  //�ֱ��ʾ Ҫ���Ƶ�vector��ͷ, Ҫ���Ƶ�vector��β , Ŀ������
    //3.2.2 ִ��cuda�Դ濽������
    hipMemcpy(cudaTestArrayPiece,doubleArrayB ,sizeof(double) * columnSize ,hipMemcpyHostToDevice); //��test�����ݴ���
    //4. ִ�е�һ���˺���
    hipEvent_t start1,stop1;
    float elapsedTime1 = 0;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventRecord(start1,0);
    MatrixSubAndSquare<<<firstBlocksPerGrid,firstThreadsPerBlock>>>(cudaDoubleArray,cudaTestArrayPiece,cudaAfterSubAndSquareDoubleArrayResult,columnSize);
    hipEventRecord(stop1,0);
    hipEventSynchronize(stop1);
    hipEventElapsedTime(&elapsedTime1,start1,stop1);
//    cout<<"first kennel function cost time:"<<elapsedTime1<<endl;

    //�������������  ��һ�����м䲽�� ���Ե�ʱ���Ŵ���
//    hipMemcpy(doubleArrayResult,cudaAfterSubAndSquareDoubleArrayResult,trainDataSize * columnSize *sizeof(double)  ,hipMemcpyDeviceToHost); //��ѵ���������ݿ��뵽�Դ���)
//    //��ӡ������м��� �м䲽�� ����ʹ��
//    for(int i=0;i<trainDataSize;i++){
//        for(int j=0;j<columnSize;j++){
//            cout<< doubleArrayResult[i*columnSize + j]<<"    ";
//        }
//        cout<<endl;
//    }
    //5. �ͷ�һ�����Դ���ڴ�  ע��û���ͷ� cudaAfterSubAndSquareDoubleArrayResult ��Ϊ�м�������Ҫʹ��
    //5.1 �ͷ��Դ�
    hipFree(cudaDoubleArray); //�ͷ� ��ά����(�Ų�Ϊһά) ѵ��������
    hipFree(cudaTestArrayPiece); //�ͷ� һά���� ���Լ���һ������
    //5.2 �ͷ��ڴ�
    free(doubleArrayA);
    free(doubleArrayB);
    //6 Ϊִ�еڶ����˺���׼���ռ�(�ڴ���Դ�)
    auto *distanceArray = new double[trainDataSize]; //�����ڴ�ռ� ������ž�������
    double *cudaDistanceArray; //����ռ� ӳ���Դ�ռ� ������ž�������
    hipMalloc((void**)&cudaDistanceArray,sizeof(double) * trainDataSize ); //�����ž����Դ�ռ�
    //7 ִ�еڶ����˺���
    hipEvent_t start2,stop2;
    float elapsedTime2 = 0;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipEventRecord(start2,0);
    sumMatrix<<<secondBlocksPerGrid,secondThreadsPerBlock>>>(cudaAfterSubAndSquareDoubleArrayResult,cudaDistanceArray,columnSize);
    hipEventRecord(stop2,0);
    hipEventSynchronize(stop2);
    hipEventElapsedTime(&elapsedTime2,start2,stop2);
//    cout<<"second kennel function cost time:"<<elapsedTime2<<endl;

//    cout<<"two kennel function cost time:"<<elapsedTime1+elapsedTime2<<endl;
    allKernelFunctionCostTime += elapsedTime1+elapsedTime2;
    //8 �����ľ������鿽�����ڴ� �Ա���ʹ��
    hipMemcpy(distanceArray,cudaDistanceArray,sizeof(double) *trainDataSize ,hipMemcpyDeviceToHost);
    //9 �ͷŵ������Դ� ��Ϊ�Ժ��ò�����
    hipFree(cudaAfterSubAndSquareDoubleArrayResult);  //�ͷŵ��м���������
    hipFree(cudaDistanceArray);  //�ͷŵ���������
//    for(int i=0;i<trainDataSize ;i++ ){  //��ӡ��������
//        cout<<distanceArray[i]<<endl;
//    }
    //10 �������ݴ���
    //10.1 ��ʼ��һЩstl �Ժ���õ�
    auto *realityAndDistanceMap = new multimap<double,string>;  //������Ϊkey ��ʵֵΪvalue �������ĺô����Զ����� ��Ҫ����multimap ��Ȼ����һ�㲻��һ�� ���Ǿ�������
    auto *weightMap = new map<string,double>;  //Ȩ��map   ���double���ݿ�����Ϊ���� key�������ظ� ���Է����� map
    set<string>::iterator setItr ;  //�����������н��(resultSet) ���set�����˽���������п��� �����ж��Ƿ�ò������ݼ� ֻ�еò����߲��ò����� �Ѿ����ź����
    map<double,string>::iterator mapIter; //���ڱ���
    //10.2 ����þ���֮����Ҫ�����ݺ���ʵֵ��Ӧ����  ֮������ͳ��Ȩֵ��ʱ����õ� realityAndDistanceMap multimap<double,string>
    for(int i=0;i<trainDataSize;i++){
        realityAndDistanceMap->insert(pair<double,string>(distanceArray[i] ,resultVector->at(i)));
    }
    free(distanceArray); //˳�ְ�distanceArray�ͷŵ� �Ժ󲻻�������
    //10.3.��ʼ��Ȩ��map ��value�����0  ͳ��Ȩ�ص�Ŀ�����ۺϿ���k������ĵ��Ӱ�� Լ�ӽ��ĵ�Ȩ��Խ��
    setItr = resultSet->begin();  //resultSet set<string> �ĵ�����
    for(;setItr!=resultSet->end();setItr++){
        weightMap->insert(pair<string,double>(*setItr,0));
    }
    //10.4 ����ǰk������ĵ���ܾ���sum ������Ȩ��
    mapIter = realityAndDistanceMap->begin();
    for(int i=0;i<k;i++,mapIter++){ //����sumֵ �������sumֵ
        sum  += mapIter->first;
    }
    //10.5 �ֱ����ǰk�����Ȩֵ ��������ʵֵ �ӵ����п��ܵ�ֵ�� ����ò�Ȩ��5.4 ���ò�Ȩ��Ϊ 8.4 ���Կ����жϴ�����ǲ��ò�
    mapIter = realityAndDistanceMap->begin();
    for(int i=0;i<k;i++ ,mapIter++){  //ǰk��Ԫ�ص�Ȩ�������
        (*weightMap)[mapIter->second] += 1 - (mapIter->first / sum) ; //����Խ�� Ȩ��Խ��  ע�������� 1- xxx
    }
    //10.6 �ҵ���ߵ��Ǹ�Ȩ�ص�ֵ �����ǲ��ò� Ȼ������ֵ��maxWeightStr
    for(pair<string,double> p :*weightMap ){
        if(p.second > maxWeight){
            maxWeight = p.second;
            maxWeightStr = p.first;
        }
    }
    //10.7 ���ݲ��Լ�����ʵֵ��Ԥ��ֵ�Ա��Ƿ�һ��  ���һ��˵��Ԥ��ɹ�
    if(resultVector->at(position).compare(maxWeightStr)  == 0)
        flag = true;  //flag ��߻�return��ȥ
    else
        flag = false;
    //11 �����ͷŵ����е��ڴ�(�Դ��Ѿ�ȫ���ͷ�)
    free(realityAndDistanceMap); //�ͷ���ʵֵ�;���map
    free(weightMap); //�ͷ�Ȩ��map
    return flag; //����Ԥ��������ʵֵ�Ƿ�ƥ��
}


int main(int argc,char * argv[]) {
    clock_t allTimeBegin = clock();
    if(argc > 1){
        fileName = argv[1];
        cout<<"����������� csv�ļ�Ϊ :   "<<argv[1]<<endl<<endl;
    }else{
        cout<<"δ������������� Ĭ��csv�ļ���:"<<fileName<<endl;
    }

    //1. ��ʼ�����в���
    vector<vector<double>> *doubleDataVector = nullptr; //��ά���� �������ѵ�����Ͳ��Լ�����������
    vector<string> * resultVector = nullptr;  //����� ���ȫ���ַ���
    set<string> * realitySet = nullptr;  //���ظ������н����
    ifstream inputFile;   //�����ļ�������

    //2.����i/o�� ���ļ� Ȼ�����yhnCsv������ȡcsv�ļ�������
    inputFile.open(fileName);  //���ļ�
    Csv * csvReader = new Csv(&inputFile);  //���ļ��������ȥ
    //3. ��ȡ����
    //3.1 ������߼��ϵĻ�ȡ
    realitySet = csvReader->getResultSet();  //��ȡ���ظ������ݼ�
    resultVector = csvReader->getResultVector(); //�õ����еĽ��
    doubleDataVector = csvReader->getDoubleData(); //�����е�ѵ�����Ͳ��Լ��������õ�

    //3.2 ��ȡȫ�ֱ���
    columnSize = doubleDataVector->at(0).size();  //�����е�����
    dataSize = doubleDataVector->size();  //��¼�����ݼ�������
    trainDataSize = trainDataProportion * dataSize;  //��¼ѵ����������  �������� �����ݼ�������
    testDataSize = dataSize - trainDataSize; //��¼���Լ�������
    //4. ͬʱ��ȫ�����ݼ��ͽ�����ݼ�������� ������������ͬ ��ô�ͽ���  ������ͬʱ���ж������Ľ��� swap�����ܺ�ʹ
    srand((unsigned int)time(NULL));  //��ʱ��Ϊ��׼�������
    for (int i = 0; i < dataSize; ++i) {  //��ཻ�� �����ݼ���С ��ʵһ����� ��������ν��
        int n1 = (rand() % dataSize);//����n���ڵ������  n������Ԫ�ظ���
        int n2 = (rand() % dataSize);
        if (n1 != n2) { //�������������� ���±�Ϊ�����������������н���
            swap(doubleDataVector->at(n1),doubleDataVector->at(n2));
            swap(resultVector->at(n1),resultVector->at(n2));
        }
    }
    csvReader->printHeaderVector();  //��ӡͷ�������ַ��� ����������е�����
    csvReader->printResultInformation(); //��ӡresult��Ϣ
    cout<<"kֵΪ��"<<k<<",�����ݼ���"<<dataSize<<"����"<<"ѵ������"<<trainDataSize<<"��,"<<"���Լ���"<<testDataSize<<"��"<<endl;
    int count = 0; //����ͳ�Ƴɹ�Ԥ�������
    for(int i=0;i<testDataSize;i++){
        bool flag =  knn(&doubleDataVector->at(trainDataSize + i), //���Լ���һ��
                         trainDataSize +i  , //���Լ���λ��
                         doubleDataVector, // ���е�����
                         resultVector, //��������ݼ�
                         realitySet); //��������п���ֵ �� set(���ظ�)
//        bool flag =  knn(&doubleDataVector->at(69), //���Լ���һ��
//                         69 , //���Լ���λ��
//                         doubleDataVector, // ���е�����
//                         resultVector, //��������ݼ�
//                         realitySet); //��������п���ֵ �� set(���ظ�)
        if(flag){
            count++;
        }
    }
    clock_t allTimeEnd = clock();

    cout<<"�˴���������ݼ���׼ȷ��Ϊ: "<<(float )count/testDataSize *100 <<"%"<<endl ;
    cout<<"all Kernel Function Cost Time: "<<allKernelFunctionCostTime<<" ms"<<endl;
    cout<<"total Cost Time:"<<(allTimeEnd - allTimeBegin)/CLOCKS_PER_SEC <<" s"<<endl;
    free(csvReader);
}
