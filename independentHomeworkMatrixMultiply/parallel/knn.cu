#include <iostream>
#include <cmath>
#include <fstream>
#include <hip/hip_runtime.h>
using namespace std;

// 训练集的大小
const int train_col = 8;
const int train_row = 614;
// 测试集的大小
const int test_col = 8;
const int test_row = 154;
// block中线程排列
const int matSub_x = 2;
const int matSub_y = 2;
const int distance_x = train_row;
const int distance_y = 1;
// 预测结果
double prediction[test_row];
__global__ void matSub(double **A, double **B, double **C) {
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    double tmp = A[row][col] - B[row][col];
    C[row][col] = tmp * tmp;
}
__global__ void distance(double **C, double *D) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    double tmp = 0;
    for (int k = 0; k < train_col; k++) {
        tmp += C[i][k];
    }
    D[i] = sqrt(tmp);
}

int main() {
    // 加载数据
    ifstream x_train_in("X_train.csv");
    ifstream y_train_in("Y_train.csv");
    ifstream x_test_in("X_test.csv");
    ifstream y_test_in("Y_test.csv");
    // 训练集
    double **x_train = new double*[train_row];
    for (int i = 0; i < train_row; i++) {
        x_train[i] = new double[train_col];
    }
    double *y_train = new double[train_row];
    // 测试集
    double **x_test = new double*[test_row];
    for (int i = 0; i < test_row; i++) {
        x_test[i] = new double[test_col];
    }
    double *y_test = new double[test_row];
    for (int i = 0; i < train_row; i++) {
        for (int j = 0; j < train_col; j++) {
            x_train_in >> x_train[i][j];
        }
    }
    for (int i = 0; i < train_row; i++) {
        y_train_in >> y_train[i];
    }
    for (int i = 0; i < test_row; i++) {
        for (int j = 0; j < test_col; j++) {
            x_test_in >> x_test[i][j];
        }
    }
    for (int i = 0; i < test_row; i++) {
        y_test_in >> y_test[i];
    }
    // ====
    double **A = new double*[train_row];
    double **B = new double*[train_row];
    double **C = new double*[train_row];
    double *D;
    double *result = new double[train_row];
    hipMalloc((void**)&D, sizeof(double)*train_row);
    for (int i = 0; i < test_row; i++) {
        for (int j = 0; j < train_row; j++) {
            hipMalloc((void**)&A[j], sizeof(double)*train_col);
            hipMalloc((void**)&B[j], sizeof(double)*train_col);
            hipMalloc((void**)&C[j], sizeof(double)*train_col);
            hipMemcpy(A[j], x_train[j], sizeof(double)*train_col, hipMemcpyHostToDevice);
            hipMemcpy(B[j], x_test[i], sizeof(double)*train_col, hipMemcpyHostToDevice);
            dim3 threadsPerBlock(matSub_x, matSub_y);
            dim3 blocksPerGrid(train_col / threadsPerBlock.x, train_row / threadsPerBlock.y);
            matSub <<< blocksPerGrid, threadsPerBlock >>> (A, B, C);
            dim3 threadsPerBlock1(distance_x, distance_y);
            dim3 blocksPerGrid1(train_row / threadsPerBlock1.x, 1);
            distance <<< blocksPerGrid1, threadsPerBlock1 >>> (C, D);
            hipMemcpy(result, D, sizeof(double)*train_row, hipMemcpyDeviceToHost);
            int index = 0;
            double minn = result[index];
            for (int k = 1; k < train_row; k++) {
                if (minn > result[k]) {
                    index = k;
                }
            }
            prediction[i] = y_train[index];
        }
    }
    int cnt = 0;
    for (int i = 0; i < train_row; i++) {
        if (prediction[i] == y_test[i]) {
            ++cnt;
        }
    }
    cout << (double)cnt / test_row << endl;
    // ====
    // 释放空间
    hipFree(D);
    delete [] result;
    for (int i = 0; i < train_row; i++) {
        delete [] x_train[i];
    }
    delete [] y_train;
    for (int i = 0; i < test_row; i++) {
        delete [] x_test[i];
    }
    delete [] y_test;
    return 0;
}