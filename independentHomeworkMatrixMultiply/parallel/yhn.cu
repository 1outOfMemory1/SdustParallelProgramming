#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
const int ND=1000;
#define size 10
using namespace std;


//int a[ND][ND],b[ND][ND],c[ND][ND];


__global__ void MatMul(int *M,int *N,int *P,int width)
{

    int Col = blockIdx.x*blockDim.x + threadIdx.x; // cloumn
    int Row = blockIdx.y*blockDim.y + threadIdx.y; // row

    float elem1 = 0.0,elem2 = 0.0,value = 0.0;
    for(int i = 0;i < width;i++)
    {
        elem1 = M[Col * width + i];//取M矩阵的一行
        elem2 = N[i * width + Row];//取N矩阵的一列
        value += elem1 * elem2;//求和
    }

    P[Col * width + Row] = value;
}


int main(int argc,char * argv[])
{
//    int hhh = atoi(argv[1]); //读取执行时参数 并把它转换为int值 这个值代表矩阵大小 size * size 大小的两个矩阵相乘
//    cout<<hhh<<endl;   // 把size打印出来
//    hipSetDevice(0);

    int (*a)[ND] = new int[ND][ND];
    int (*b)[ND] = new int[ND][ND];
    int (*c)[ND] = new int[ND][ND];

//    int *c = new int[ND*ND];
//    for(int i=0;i<ND;i++){
//        c[i] = new int[ND];
//    }

    int *M,*N,*P;

    int width = ND;
    dim3 gridSize(ND/size,ND/size);
    dim3 blockSize(size,size);

    hipEvent_t start,stop;
    float elapsedTime = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //设备端内存分配
    hipMalloc((void**)&M,ND * ND * sizeof(int));
    hipMalloc((void**)&N,ND * ND * sizeof(int));
    hipMalloc((void**)&P,ND * ND * sizeof(int));

    //初始化
    for(int i = 0;i < ND;i++)
    {
        for(int j = 0;j < ND;j++)
        {
            a[i][j] = 1;
            b[i][j] = 1;
        }
    }

    int Size = ND * ND;
    //数据拷贝，主机到设备
    hipMemcpy(M,a,Size * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(N,b,Size * sizeof(int),hipMemcpyHostToDevice);

    hipEventRecord(start,0);
    MatMul<<<gridSize,blockSize>>>(M,N,P,width);//调用核函数
    hipDeviceSynchronize();
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);
    hipMemcpy(c,P,Size * sizeof(int),hipMemcpyDeviceToHost);



    printf("cost time : %f ms $$$$ %f s \n ",elapsedTime,elapsedTime/1000);
//    for(int i=0;i<ND;i++){
//        for(int j=0;j<ND;j++){
//            printf("%d ",c[i][j]);
//        }
//    }


    //释放设备内存
    hipFree(M);
    hipFree(N);
    hipFree(P);
    free(a);
    free(b);
    free(c);
    return 0;
}
