#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include <ctime>
#include <cstdlib>
#include "yhncsv.h"
#include "common.h"

using namespace std;

// ȫ�ֱ���
double trainDataProportion = (float )2/3;  //���ڹ涨ѵ����ռ�����ݵı���
int dataSize = 0;
int trainDataSize = 0; //���ڼ�¼ѵ�����Ĵ�С
int testDataSize = 0; //���ڼ�¼���Լ��Ĵ�С
int columnSize = 0;  //���ڼ�¼�����е�����
int threadSize = 2; //һ������ �߳�����32 * 32  =1024 ���ֵ



//һ��������������   ���Լ��е�һ�� ����ѵ�����е�ÿһ����������Ȼ��ƽ��
__global__ void MatrixSubAndSquare(double *trainSet,  //�����ά���� ÿһ��������
                                   double *oneRowOftestSet, //��Ҫ���������Լ���ĳһ��
                                   double *afterSubAndSquareResultArray, //�������������ƽ���������м�����
                                   int columnSize
                                   ){
    //�����grid �ֲ�  rowSize/ThreadSize columnSize/ThreadSize   ThreadSize=32 ThreadSize=32
    // rowSize/ThreadSize * ThreadSize = rowSize   columnSize/ThreadSize = columnSize
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int col = blockDim.y * blockIdx.y + threadIdx.y;
    double value = trainSet[row * columnSize + col]  - oneRowOftestSet[col]; //��ƽ�����ֵ������������
    afterSubAndSquareResultArray[row * columnSize + col] = value * value;
}

__global__ void knn(){

}


//��Ϊvector�������洢���ݵ� ����ֱ�ӽ����ڴ濽������
//memcpy(doubleArray,&doubleVector[0],doubleVectorSize * sizeof(double));


int myRandom(int i){
    int randomNum = rand() % i;
    return randomNum;
}
//double distance(vector<double>* d1,vector<double> * d2){
//    double dis = 0;
//    int size = d1->size();
//    for (int i = 0;i < size;i++){
//        dis += pow((d1->at(i) - d2->at(i)),2);
//    }
//    return pow(dis,0.5);
//}


//���knn������һ������ �������  һ�в������� ����  ����ѵ���������еľ��� Ȼ����������k��������Ԥ��ֵ
bool knn(vector<double> * testPiece, vector<vector<double>> *doubleDataArray){
    double maxWeight = -1;  // �����洢���Ȩ��
    string maxWeightStr = "";  // �����洢���Ȩ�ص��ַ���  Ҳ����Ԥ��ֵ
    bool flag = false;   // ���ظ������� �����ж�Ԥ���Ƿ���ȷ
    double sum = 0;
    auto *distanceVector = new vector<double>;

    // 1.�����
    dim3 firstBlocksPerGrid(trainDataSize/threadSize,columnSize/threadSize);
    dim3 firstThreadsPerBlock(threadSize,threadSize);
    //����ռ�
    double * result1 = new double[trainDataSize*columnSize];

    double *cudaDoubleArray; //��������
    double * cudaTestArrayPiece; //һ�в��Լ�����
    double *cudaAfterSubAndSquareDoubleArrayResult;

    hipMalloc((void**)&cudaDoubleArray,sizeof(double) * trainDataSize * columnSize ); //�����Դ�ռ�
    hipMalloc((void**)&cudaTestArrayPiece,sizeof(double) * columnSize);  //����һά����Ŀռ�
    hipMalloc((void**)&cudaAfterSubAndSquareDoubleArrayResult,sizeof(double) * trainDataSize * columnSize ); //�����Դ�ռ�

    double *xxx = new double[trainDataSize * columnSize];


    for(int i=0;i<trainDataSize;i++){
        for(int j =0;j<columnSize;j++){
            xxx[i* columnSize +j] = doubleDataArray->at(i).at(j);
//            cout<<xxx[i* columnSize +j]<<"  ";
        }
//        cout<<endl;
    }



//    for(int i=0;i<trainDataSize;i++){
//        hipMemcpy(&cudaDoubleArray[i],&doubleDataArray->at(i),
//                   sizeof(double)  * columnSize ,
//                   hipMemcpyHostToDevice); //��ѵ���������ݿ��뵽�Դ���
//    }

    hipMemcpy(cudaDoubleArray,xxx,
                   sizeof(double)  * columnSize * trainDataSize ,
                   hipMemcpyHostToDevice); //��ѵ���������ݿ��뵽�Դ���
    double  *yyy = new double[columnSize];
    for(int i=0;i<columnSize;i++){
        yyy[i] = testPiece->at(i);
//        cout<<yyy[i]<<"  ";
    }
    hipMemcpy(cudaTestArrayPiece,yyy,sizeof(double) * columnSize ,hipMemcpyHostToDevice); //��test�����ݴ���


/*
 double *trainSet,  //�����ά���� ÿһ��������
 double *oneRowOftestSet, //��Ҫ���������Լ���ĳһ��
 double *afterSubAndSquareResultArray //�������������ƽ���������м�����
 * */


    //ִ�к˺���
    MatrixSubAndSquare<<<firstBlocksPerGrid,firstThreadsPerBlock>>>(cudaDoubleArray,cudaTestArrayPiece,cudaAfterSubAndSquareDoubleArrayResult,columnSize);
//    MatrixSubAndSquare<<<1,dim3(1,1000)>>>(cudaDoubleArray,cudaTestArrayPiece,cudaAfterSubAndSquareDoubleArrayResult,columnSize);

    //�������������
    hipMemcpy(result1,cudaAfterSubAndSquareDoubleArrayResult,
                      trainDataSize * columnSize *sizeof(double)  ,
                      hipMemcpyDeviceToHost); //��ѵ���������ݿ��뵽�Դ���)


//    for(int i=0;i<trainDataSize;i++){
//        for(int j=0;j<columnSize;j++){
//            cout<< result1[i*columnSize + j]<<"    ";
//        }
//        cout<<endl;
//    }
//
    for(int i=0;i<trainDataSize;i++){
        yyy[i] = 0;
        for(int j=0;j<columnSize;j++){
            yyy[i] += result1[i*columnSize + j];
        }
        yyy[i] = sqrt(yyy[i]);
    }


    sort(yyy,yyy+trainDataSize);
    for(int i =0; i< trainDataSize;i++){
        cout<<yyy[i]<<endl;
    }
    int bbbb = 0;



    // 2.������������
    // 3.ȡǰk��
    // 4.��Ȩƽ��


    return false;
}


int main() {
    srand(time(0));   //��������� ����ʱ�����������
    int k=10; //  ��������ȡǰ k ���������������
    //��һ�� ��ʼ�����в���
    vector<vector<double>> *doubleDataArray = nullptr; //��ά���� �����������
    vector<string> * headerNameVector = nullptr;
    ifstream inputFile;   //�����ļ�������
    string fileName = "../KNN_Data.csv";  //�����ļ�����
    inputFile.open(fileName);  //���ļ�
    Csv * csvReader = new Csv(&inputFile);  //���ļ��������ȥ
    headerNameVector =  csvReader->getHeaderNameVector(); //��ȡͷ�������� ���������������
    doubleDataArray = csvReader->getDoubleData(); //�����е������õ�
    columnSize = doubleDataArray->at(0).size();  //�����е�����
    dataSize = doubleDataArray->size();  //��¼���ݼ�������
    trainDataSize = trainDataProportion * dataSize;  //��¼ѵ����������
    testDataSize = (1-trainDataProportion) * dataSize; //��¼���Լ�������
    random_shuffle(doubleDataArray->begin(),doubleDataArray->end(),myRandom); // �����ݴ���ע����������� myRandom��һ��������ַ ��random_shuffle���� �������
    knn(&doubleDataArray->at(trainDataSize),doubleDataArray);


    //    csvReader->printDoubleDataVector(); //��ӡ��������
//    csvReader->printHeaderVector();  //��ӡͷ�������ַ��� ����������е�����
//    csvReader->printResultVector(); //��ӡ����е�����
//    csvReader->printResultInformation();




}
