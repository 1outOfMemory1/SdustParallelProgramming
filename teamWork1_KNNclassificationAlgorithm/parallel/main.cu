#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include <ctime>
#include <cstdlib>
#include "yhncsv.h"
#include "common.h"
#include <map>

using namespace std;

// ȫ�ֱ���
double trainDataProportion = (float )2/3;  //���ڹ涨ѵ����ռ�����ݵı���
int dataSize = 0;
int trainDataSize = 0; //���ڼ�¼ѵ�����Ĵ�С
int testDataSize = 0; //���ڼ�¼���Լ��Ĵ�С
int columnSize = 0;  //���ڼ�¼�����е�����
int threadSize = 2; //һ������ �߳�����32 * 32  =1024 ���ֵ



//һ��������������   ���Լ��е�һ�� ����ѵ�����е�ÿһ����������Ȼ��ƽ��
__global__ void MatrixSubAndSquare(double *trainSet,  //�����ά���� ÿһ��������
                                   double *oneRowOftestSet, //��Ҫ���������Լ���ĳһ��
                                   double *afterSubAndSquareResultArray, //�������������ƽ���������м�����
                                   int columnSize
                                   ){
    //�����grid �ֲ�  rowSize/ThreadSize columnSize/ThreadSize   ThreadSize=32 ThreadSize=32
    // rowSize/ThreadSize * ThreadSize = rowSize   columnSize/ThreadSize = columnSize
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int col = blockDim.y * blockIdx.y + threadIdx.y;
    double value = trainSet[row * columnSize + col]  - oneRowOftestSet[col]; //��ƽ�����ֵ������������
    afterSubAndSquareResultArray[row * columnSize + col] = value * value;
}

int myRandom(int i){
    int randomNum = rand() % i;
    return randomNum;
}
__global__ void sumMatrix(double *aa,double *distance,int columnSize){
    int x = blockIdx.x *blockDim.x + threadIdx.x;
    double value = 0;
    for(int i=0;i<columnSize;i++){
        value += aa[x * columnSize + i];
    }
    distance[x] = sqrt(value);
}


//���knn������һ������ �������  һ�в������� ����  ����ѵ���������еľ��� Ȼ����������k��������Ԥ��ֵ
bool knn(vector<double> * testPiece, int position ,vector<vector<double>> *doubleDataArray,vector<string> * resultVector,int k,set<string>* resultSet){
    double maxWeight = -1;  // �����洢���Ȩ��
    string maxWeightStr = "";  // �����洢���Ȩ�ص��ַ���  Ҳ����Ԥ��ֵ
    bool flag = false;   // ���ظ������� �����ж�Ԥ���Ƿ���ȷ
    double sum = 0;

    // 1.�����
    dim3 firstBlocksPerGrid(trainDataSize/threadSize,columnSize/threadSize);
    dim3 firstThreadsPerBlock(threadSize,threadSize);
    //����ռ�
    double * doubleArrayResult = new double[trainDataSize*columnSize];
    double *cudaDoubleArray; //��������
    double *cudaTestArrayPiece; //һ�в��Լ�����
    double *cudaAfterSubAndSquareDoubleArrayResult;
    hipMalloc((void**)&cudaDoubleArray,sizeof(double) * trainDataSize * columnSize ); //�����Դ�ռ�
    hipMalloc((void**)&cudaTestArrayPiece,sizeof(double) * columnSize);  //����һά����Ŀռ�
    hipMalloc((void**)&cudaAfterSubAndSquareDoubleArrayResult,sizeof(double) * trainDataSize * columnSize ); //�����Դ�ռ�
    double *doubleArrayA = new double[trainDataSize * columnSize]; //��ֵ�����ֵ������ vector����
    for(int i=0;i<trainDataSize;i++){
        for(int j =0;j<columnSize;j++){
            doubleArrayA[i* columnSize +j] = doubleDataArray->at(i).at(j);
        }
    }
    hipMemcpy(cudaDoubleArray,doubleArrayA,sizeof(double)  * columnSize * trainDataSize ,hipMemcpyHostToDevice); //��ѵ���������ݿ��뵽�Դ���
    double  *doubleArrayB = new double[columnSize];
    for(int i=0;i<columnSize;i++){
        doubleArrayB[i] = testPiece->at(i);
    }
    hipMemcpy(cudaTestArrayPiece,doubleArrayB ,sizeof(double) * columnSize ,hipMemcpyHostToDevice); //��test�����ݴ���
    //ִ�к˺���
    MatrixSubAndSquare<<<firstBlocksPerGrid,firstThreadsPerBlock>>>(cudaDoubleArray,cudaTestArrayPiece,cudaAfterSubAndSquareDoubleArrayResult,columnSize);
    //�������������  ��һ�����м䲽��
//    hipMemcpy(doubleArrayResult,cudaAfterSubAndSquareDoubleArrayResult,trainDataSize * columnSize *sizeof(double)  ,hipMemcpyDeviceToHost); //��ѵ���������ݿ��뵽�Դ���)
//    //��ӡ���
//    for(int i=0;i<trainDataSize;i++){
//        for(int j=0;j<columnSize;j++){
//            cout<< doubleArrayResult[i*columnSize + j]<<"    ";
//        }
//        cout<<endl;
//    }



    //�ͷ��Դ�  ע��û���ͷ� size�� columnSize * trainDataSize ���Դ� ��Ϊ��߻���Ҫʹ��
    hipFree(cudaDoubleArray);
    hipFree(cudaTestArrayPiece);
//    hipFree(cudaAfterSubAndSquareDoubleArrayResult);  //ע�����ﲻ�����ͷ���Ϊ��������һ������
    //�ͷ��ڴ�
    free(doubleArrayA);
    free(doubleArrayB);

    double *distanceArray = new double[trainDataSize]; //����ռ� ������ž�������
    double *cudaDistanceArray; //����ռ� ������ž�������
    hipMalloc((void**)&cudaDistanceArray,sizeof(double) * trainDataSize ); //�����ž����Դ�ռ�
    sumMatrix<<<dim3(trainDataSize/threadSize),dim3(threadSize)>>>(cudaAfterSubAndSquareDoubleArrayResult,cudaDistanceArray,columnSize);
    hipMemcpy(distanceArray,cudaDistanceArray,sizeof(double) *trainDataSize ,hipMemcpyDeviceToHost);

//    for(int i=0;i<trainDataSize ;i++ ){
//        cout<<distanceArray[i]<<endl;
//    }


    //����þ���֮����Ҫ�����ݺ���ʵֵ��Ӧ����
     auto *realityAndDistanceMap = new map<double,string>;
     for(int i=0;i<trainDataSize;i++){
         realityAndDistanceMap->insert(pair<double,string>(distanceArray[i] ,resultVector->at(i)));
     }


    // 2.������������
    // 3.ȡǰk��


    // 4.ͳ��Ȩ�� �������׼ȷ
    auto *weightMap = new map<string,double>;
    auto  setItr = resultSet->begin();
    for(;setItr!=resultSet->end();setItr++){
        weightMap->insert(pair<string,double>(*setItr,0));
    }
    map<double,string>::iterator iter;
    iter = realityAndDistanceMap->begin();
    for(int i=0;i<k;i++,iter++){ //����sumֵ �������sumֵ
        sum  += iter->first;
    }

    iter = realityAndDistanceMap->begin();
    for(int i=0;i<k;i++ ,iter++){  //ǰk��Ԫ�ص�Ȩ�������
        (*weightMap)[iter->second] += 1 - (iter->first / sum) ; //����Խ�� Ȩ��Խ��
    }



    for(pair<string,double> p :*weightMap ){
        if(p.second > maxWeight){
            maxWeight = p.second;
            maxWeightStr = p.first;
        }
    }

//    if(resultVector->at(position) == realityAndDistanceMap->begin()->second){
//        cout<<"Ԥ����ȷ"<<endl;
//        return true;
//    }
    if(resultVector->at(position).compare(maxWeightStr)  == 0){
//        cout<<"Ԥ����ȷ"<<endl;
        return true;
    }
    else{
//        cout<<"Ԥ�����"<<endl;
        return false;
    }
}


int main() {
    int k=10; //  ��������ȡǰ k ���������������
    //��һ�� ��ʼ�����в���
    vector<vector<double>> *doubleDataArray = nullptr; //��ά���� �����������
    vector<string> * headerNameVector = nullptr;
    vector<string> * resultVector = nullptr;
    set<string> * realitySet = nullptr;
    ifstream inputFile;   //�����ļ�������
    string fileName = "../diabetes.csv";  //�����ļ�����
    inputFile.open(fileName);  //���ļ�
    Csv * csvReader = new Csv(&inputFile);  //���ļ��������ȥ
    realitySet = csvReader->getResultSet();
    headerNameVector =  csvReader->getHeaderNameVector(); //��ȡͷ�������� ���������������
    doubleDataArray = csvReader->getDoubleData(); //�����е������õ�
//    csvReader->printDoubleDataVector(); //��ӡ��������
    resultVector = csvReader->getResultVector();
    columnSize = doubleDataArray->at(0).size();  //�����е�����
    dataSize = doubleDataArray->size();  //��¼���ݼ�������
    trainDataSize = trainDataProportion * dataSize;  //��¼ѵ����������
    testDataSize = dataSize - trainDataSize; //��¼���Լ�������
    //random_shuffle(doubleDataArray->begin(),doubleDataArray->end(),myRandom); // �����ݴ���ע����������� myRandom��һ��������ַ ��random_shuffle���� �������
    //������� ������������ͬ ��ô�ͽ��� ���� ��
    srand((unsigned int)time(NULL));
    for (int i = 0; i < dataSize; ++i) {
        int n1 = (rand() % dataSize);//����n���ڵ������  n������Ԫ�ظ���
        int n2 = (rand() % dataSize);
        if (n1 != n2) { //�������������� ���±�Ϊ�����������������н���
            swap(doubleDataArray->at(n1),doubleDataArray->at(n2));

            swap(resultVector->at(n1),resultVector->at(n2));
        }
    }


    int count = 0;
    for(int i=0;i<testDataSize;i++){
        bool flag =  knn(&doubleDataArray->at(trainDataSize - 1 + i),trainDataSize +i -1 ,doubleDataArray,resultVector,k,realitySet);
        if(flag){
            count++;
        }
    }
    cout<<"׼ȷ��Ϊ"<<(float )count/testDataSize *100 <<"%" ;
//    csvReader->printHeaderVector();  //��ӡͷ�������ַ��� ����������е�����
//    csvReader->printResultVector(); //��ӡ����е�����
//    csvReader->printResultInformation();




}
