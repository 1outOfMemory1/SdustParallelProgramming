
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;


/*�����������������һ��
�������������һ�е��ۼ�ֵ Ȼ����п���
 * */

//__global__ void sub(double *aa,double *bb,double *result){
//    int row = blockDim.x * blockIdx.x + threadIdx.x;
//    int col = blockDim.y * blockIdx.y + threadIdx.y;
//    result[row *10 +col] = aa[row *10 +col] - bb[col];
//}

//__global__ void sum(double *aa,double *distance){
//    int y =  threadIdx.y;
//    double value = 0;
//    for(int i=0;i<10;i++){
//        value += aa[y * 10 + i];
//    }
//    distance[y] = sqrt(value);
//}

__global__ void sum(double *aa,double *distance){
    int x = blockIdx.x *blockDim.x + threadIdx.x;
    double value = 0;
    for(int i=0;i<10;i++){  //�ۼ�һ���е�����
        value += aa[x * 10 + i];
    }
    distance[x] = sqrt(value); //��sum���п���
}

int main() {
    double *aa = new double[600];
    double *distance = new double[60];

    for(int i=0;i<600;i++){
        aa[i] = 1;
    }

    for(int j=20;j<30;j++){
        aa[j] = 2;
    }//��֤�ɹ�
    double *cudaAA;
    double *cudaDistance;
    hipMalloc((void**)&cudaAA,sizeof(double) *600);
    hipMalloc((void**)&cudaDistance,sizeof(double) * 60);

    hipMemcpy(cudaAA,aa,sizeof(double) * 600,hipMemcpyHostToDevice);

    sum<<<dim3(3),dim3(20)>>>(cudaAA,cudaDistance);
    hipMemcpy(distance,cudaDistance,sizeof(double) * 60,hipMemcpyDeviceToHost);
    for(int i=0;i<60;i++){
        cout<<distance[i]<<endl;
    }
}
