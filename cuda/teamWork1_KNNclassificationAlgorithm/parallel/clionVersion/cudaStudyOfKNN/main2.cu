
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;


/*这个函数跟上两个不一样
这个是用来计算一行的累加值 然后进行开方
 * */

//__global__ void sub(double *aa,double *bb,double *result){
//    int row = blockDim.x * blockIdx.x + threadIdx.x;
//    int col = blockDim.y * blockIdx.y + threadIdx.y;
//    result[row *10 +col] = aa[row *10 +col] - bb[col];
//}

//__global__ void sum(double *aa,double *distance){
//    int y =  threadIdx.y;
//    double value = 0;
//    for(int i=0;i<10;i++){
//        value += aa[y * 10 + i];
//    }
//    distance[y] = sqrt(value);
//}

__global__ void sum(double *aa,double *distance){
    int x = blockIdx.x *blockDim.x + threadIdx.x;
    double value = 0;
    for(int i=0;i<10;i++){  //累加一整行的数据
        value += aa[x * 10 + i];
    }
    distance[x] = sqrt(value); //将sum进行开方
}

int main() {
    double *aa = new double[600];
    double *distance = new double[60];

    for(int i=0;i<600;i++){
        aa[i] = 1;
    }

    for(int j=20;j<30;j++){
        aa[j] = 2;
    }//验证成功
    double *cudaAA;
    double *cudaDistance;
    hipMalloc((void**)&cudaAA,sizeof(double) *600);
    hipMalloc((void**)&cudaDistance,sizeof(double) * 60);

    hipMemcpy(cudaAA,aa,sizeof(double) * 600,hipMemcpyHostToDevice);

    sum<<<dim3(3),dim3(20)>>>(cudaAA,cudaDistance);
    hipMemcpy(distance,cudaDistance,sizeof(double) * 60,hipMemcpyDeviceToHost);
    for(int i=0;i<60;i++){
        cout<<distance[i]<<endl;
    }
}
