#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
int arrayScale = 2000; //设置矩阵规模 全局变量
int arrayScale_square = arrayScale * arrayScale;  //算出矩阵规模的平方 之后的程序会用到
#define size 2  //这个值是更改一个块中有多少个线程的  我设置的是二维的thread排布 10 *10 为100 < 1024 因为老师给的数据都是10的倍数 所以设置10很合适
using namespace std;


__global__ void MatMul(int* M, int* N, int* P, int scale)  //真正的核心函数 传入显存中的A B数组 result数组 和 数组规模
{
    //其实并行程序设计的目的就是同时计算 如果你的数组规模是10 * 10  那么需要用到100个线程计算result矩阵的每一个值
    //所以并行程序的核心是定位到这100个线程 (多维降维到二维或者一维) 然后把计算后的信息存入到显存中
    int Col = blockIdx.x * blockDim.x + threadIdx.x; // cloumn 这里是将4维 降维到 2维  去除block的边框就做到了 这一行是定位到那一列
    int Row = blockIdx.y * blockDim.y + threadIdx.y; // row   这一行是定位到哪一个行
    float elem1 = 0.0, elem2 = 0.0, value = 0.0;
    for (int i = 0; i < scale; i++)
    {
        elem1 = M[Row * scale + i];//取M矩阵的一行
        elem2 = N[i * scale + Col];//取N矩阵的一列
        value += elem1 * elem2;//求和
    }
    P[ Row * scale + Col] = value;
}


int main(int argc,char * argv[])
{

    if(argc > 1){
        int hhh = atoi(argv[1]); //读取执行时参数 并把它转换为int值 这个值代表矩阵大小 size * size 大小的两个矩阵相乘
        arrayScale = hhh;
        arrayScale_square = arrayScale * arrayScale;
        cout<<"已输入参数， 矩阵规模为"<<arrayScale<<" * "<<arrayScale<<endl;
    }else{
        cout<<"未输入参数！！！ 默认矩阵规模为"<<arrayScale<<" * "<<arrayScale<<endl;
    }

    int *intArrayA = new int[arrayScale_square];   // A矩阵
    int *intArrayB = new int[arrayScale_square];   // B矩阵
    int *intArrayResult = new int[arrayScale_square];  // 结果矩阵

    int *gpuMappingIntArrayA,*gpuMappingIntArrayB,*gpuMappingIntArrayResult;  //显存映射矩阵

    dim3 blocksPerGrid(arrayScale/size,arrayScale/size);  // grid中block排布方式
    dim3 threadsPerBock(size,size);  // block中thread的排布方式
    hipEvent_t start,stop;  // 记录cuda的运行时间
    float elapsedTime = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //cuda中申请矩阵A B和结果矩阵的空间
    hipMalloc((void**)&gpuMappingIntArrayA,arrayScale_square * sizeof(int));
    hipMalloc((void**)&gpuMappingIntArrayB,arrayScale_square * sizeof(int));
    hipMalloc((void**)&gpuMappingIntArrayResult,arrayScale_square * sizeof(int));


    //初始化 A B数组
    for(int i = 0;i < arrayScale;i++)
    {
        for(int j = 0;j < arrayScale;j++)
        {
            intArrayA[i*arrayScale + j] = 1;
            intArrayB[i*arrayScale + j] = 1;
        }
    }
//    intarraya[2] = 10;
//    intarraya[3] = 3;
//    intarrayb[3] = 1;


    //数据拷贝，主机到设备  将内存中的 A B 数组数据拷贝到 显存中的A B数组中去
    hipMemcpy(gpuMappingIntArrayA,intArrayA,arrayScale_square * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(gpuMappingIntArrayB,intArrayB,arrayScale_square * sizeof(int),hipMemcpyHostToDevice);

    hipEventRecord(start,0);
    // 执行核函数 计算结果数组的每一个值
    MatMul<<<blocksPerGrid,threadsPerBock>>>(gpuMappingIntArrayA,gpuMappingIntArrayB,gpuMappingIntArrayResult,arrayScale);//调用核函数
    hipDeviceSynchronize();
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);
    // 将结果数组的每一个值拷贝回内存
    hipMemcpy(intArrayResult,gpuMappingIntArrayResult,arrayScale_square * sizeof(int),hipMemcpyDeviceToHost);


    // 输出执行cuda执行时间
    printf("cost time : %f ms $$$$ %f s \n",elapsedTime,elapsedTime/1000);
//    for(int i=0;i<arrayScale;i++){
//        for(int j=0;j<arrayScale;j++){
//            printf("%d ",intArrayResult[i*arrayScale + j]);
//        }
//        printf("\n");
//    }


    //释放设备内存
    hipFree(gpuMappingIntArrayA);
    hipFree(gpuMappingIntArrayB);
    hipFree(gpuMappingIntArrayResult);
    free(intArrayA);
    free(intArrayB);
    free(intArrayResult);
    return 0;
}
