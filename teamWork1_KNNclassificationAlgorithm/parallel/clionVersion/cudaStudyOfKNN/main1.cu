
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

/*
这个程序是在main.cu 的基础上将block和thread改为二维排布 然后再进行平方
这样做的目的是计算出中间结果 之后再进行一下累加 然后进行开方就能算出距离了 （main2 函数中进行这个计算）

*/
__global__ void sub(double *aa,double *bb,double *result){
    //四维排布 降维为二维排布  其他还是一样的
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int col = blockDim.y * blockIdx.y + threadIdx.y;
    result[row *10 +col] =pow(aa[row *10 +col] - bb[col] , 2);
}

int main() {
    double *aa = new double[600];
    double *bb = new double[10];
    double *result = new double[600];
    for(int i=0;i<600;i++){
        aa[i] = 4;
    }
    aa[23] = 13; //用来检验数据是否正确 主要是矩阵是否是转置的 事实证明没有
    cout<<endl;
    for(int j=0;j<10;j++){
        bb[j] = 1;
    }
    double *cudaAA;
    double *cudaBB;
    double *cudaResult;
    hipMalloc((void**)&cudaAA,sizeof(double) *600);
    hipMalloc((void**)&cudaBB,sizeof(double) * 10);
    hipMalloc((void**)&cudaResult,sizeof(double) * 600);

    hipMemcpy(cudaAA,aa,sizeof(double) * 600,hipMemcpyHostToDevice);
    hipMemcpy(cudaBB,bb,sizeof(double) * 10,hipMemcpyHostToDevice);

    sub<<<dim3(6,1),dim3(10,10)>>>(cudaAA,cudaBB,cudaResult);
    hipMemcpy(result,cudaResult,sizeof(double) * 600,hipMemcpyDeviceToHost);
    for(int i=0;i<600;i++){
        if(i%10 == 0 && i!=0)
            cout<<endl;
        cout<<result[i]<<" ";
    }
}
