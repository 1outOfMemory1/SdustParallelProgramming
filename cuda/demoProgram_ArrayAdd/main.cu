#include "hip/hip_runtime.h"


#include <stdio.h>
//英伟达cuda示例程序
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    //执行add函数 进行相加 判断返回值 如果有错误输出函数执行失败的提示
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
           c[0], c[1], c[2], c[3], c[4]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    //选择一个GPU去跑程序 如果你有多个GPU的话 可以进行切换
//    cudaStatus = cudaSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "GPU选择失败 请查看选择的GPU是否正确 hipSetDevice failed!  Do you have a CUDA-capable GPU installed? \n ");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)
    //以下三块 是在GPU中申请显存空间 申请的空间是一个数组的大小
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "1 显存申请失败 hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "2 显存申请失败 hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "3 显存申请失败 hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    //以下两块是 将内存中存储的 a数组和b数组的内容拷贝到GPU显存中去
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "1 将数据从内存复制到显存失败 hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "2 将数据从内存复制到显存失败 hipMemcpy failed!");
        goto Error;
    }



    // Launch a kernel on the GPU with one thread for each element.
    //以下是真正的并行程序的执行代码 申请了一个块 里边有传入参数size个int的空间  程序的主要内容是 a b数组相加得到数组c
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    //检查在启动核心的时候是否存在错误 如果存在就报错
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "启动核心失败 addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    // 下方函数等待并行程序执行完毕 如果执行出错误 那么就结束程序 输出错误信息
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "GPU核心运算出错  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    // 这一步将已经在GPU算好的内容存入内存中的 c中以便于程序读取
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "将GPU显存中的数据存入到内存中失败 hipMemcpy failed!");
        goto Error;
    }


    //程序的最后进行内存的释放   如果上述过程出现错误也会直接跳转到这个地方进行显存的释放
    Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;  //返回最终的cuda执行情况 如果有报错 那么可以查找到报错代号
}