#include "hip/hip_runtime.h"


#include <stdio.h>
//Ӣΰ��cudaʾ������
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    //ִ��add���� ������� �жϷ���ֵ ����д����������ִ��ʧ�ܵ���ʾ
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
           c[0], c[1], c[2], c[3], c[4]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    //ѡ��һ��GPUȥ�ܳ��� ������ж��GPU�Ļ� ���Խ����л�
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "GPUѡ��ʧ�� ��鿴ѡ���GPU�Ƿ���ȷ hipSetDevice failed!  Do you have a CUDA-capable GPU installed? \n ");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)
    //�������� ����GPU�������Դ�ռ� ����Ŀռ���һ������Ĵ�С
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "1 �Դ�����ʧ�� hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "2 �Դ�����ʧ�� hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "3 �Դ�����ʧ�� hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    //���������� ���ڴ��д洢�� a�����b��������ݿ�����GPU�Դ���ȥ
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "1 �����ݴ��ڴ渴�Ƶ��Դ�ʧ�� hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "2 �����ݴ��ڴ渴�Ƶ��Դ�ʧ�� hipMemcpy failed!");
        goto Error;
    }



    // Launch a kernel on the GPU with one thread for each element.
    //�����������Ĳ��г����ִ�д��� ������һ���� ����д������size��int�Ŀռ�  �������Ҫ������ a b������ӵõ�����c
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    //������������ĵ�ʱ���Ƿ���ڴ��� ������ھͱ���
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "��������ʧ�� addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    // �·������ȴ����г���ִ����� ���ִ�г����� ��ô�ͽ������� ���������Ϣ
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "GPU�����������  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    // ��һ�����Ѿ���GPU��õ����ݴ����ڴ��е� c���Ա��ڳ����ȡ
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "��GPU�Դ��е����ݴ��뵽�ڴ���ʧ�� hipMemcpy failed!");
        goto Error;
    }


    //������������ڴ���ͷ�   ����������̳��ִ���Ҳ��ֱ����ת������ط������Դ���ͷ�
    Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;  //�������յ�cudaִ����� ����б��� ��ô���Բ��ҵ��������
}