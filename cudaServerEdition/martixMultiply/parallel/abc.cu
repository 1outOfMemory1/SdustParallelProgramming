#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
int arrayScale = 2000;
int arrayScale_square = arrayScale * arrayScale;
#define size 10
using namespace std;


__global__ void MatMul(int *M,int *N,int *P,int width)
{

    int Col = blockIdx.x*blockDim.x + threadIdx.x; // cloumn
    int Row = blockIdx.y*blockDim.y + threadIdx.y; // row
    float elem1 = 0.0,elem2 = 0.0,value = 0.0;
    for(int i = 0;i < width;i++)
    {
        elem1 = M[Col * width + i];//取M矩阵的一行
        elem2 = N[i * width + Row];//取N矩阵的一列
        value += elem1 * elem2;//求和
    }

    P[Col * width + Row] = value;
}


int main(int argc,char * argv[])
{

    if(argc > 1){ 
	int hhh = atoi(argv[1]); //读取执行时参数 并把它转换为int值 这个值代表矩阵大小 size * size 大小的两个矩阵相乘
        arrayScale = hhh;
        arrayScale_square = arrayScale * arrayScale;
        cout<<"已输入参数， 矩阵规模为"<<arrayScale<<" * "<<arrayScale<<endl;
    }else{
        cout<<"未输入参数！！！ 默认矩阵规模为"<<arrayScale<<" * "<<arrayScale<<endl;
    }
    int *intArrayA = new int[arrayScale_square];
    int *intArrayB = new int[arrayScale_square];
    int *intArrayResult = new int[arrayScale_square];

    int *gpuMappingIntArrayA,*gpuMappingIntArrayB,*gpuMappingIntArrayResult;

    dim3 blocksPerGrid(arrayScale/size,arrayScale/size);
    dim3 threadsPerBock(size,size);

    hipEvent_t start,stop;
    float elapsedTime = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //设备端内存分配

    hipMalloc((void**)&gpuMappingIntArrayA,arrayScale_square * sizeof(int));
    hipMalloc((void**)&gpuMappingIntArrayB,arrayScale_square * sizeof(int));
    hipMalloc((void**)&gpuMappingIntArrayResult,arrayScale_square * sizeof(int));


    //初始化
    for(int i = 0;i < arrayScale;i++)
    {
        for(int j = 0;j < arrayScale;j++)
        {
            intArrayA[i*arrayScale + j] = 1;
            intArrayB[i*arrayScale + j] = 1;
        }
    }

    //数据拷贝，主机到设备
    hipMemcpy(gpuMappingIntArrayA,intArrayA,arrayScale_square * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(gpuMappingIntArrayB,intArrayB,arrayScale_square * sizeof(int),hipMemcpyHostToDevice);

    hipEventRecord(start,0);
    MatMul<<<blocksPerGrid,threadsPerBock>>>(gpuMappingIntArrayA,gpuMappingIntArrayB,gpuMappingIntArrayResult,arrayScale);//调用核函数
    hipDeviceSynchronize();
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);
    hipMemcpy(intArrayResult,gpuMappingIntArrayResult,arrayScale_square * sizeof(int),hipMemcpyDeviceToHost);



    printf("cost time : %f ms $$$$ %f s \n ",elapsedTime,elapsedTime/1000);
//    for(int i=0;i<arrayScale;i++){
//        for(int j=0;j<arrayScale;j++){
//            printf("%d ",intArrayResult[i*arrayScale + j]);
//        }
//    }


    //释放设备内存
    hipFree(gpuMappingIntArrayA);
    hipFree(gpuMappingIntArrayB);
    hipFree(gpuMappingIntArrayResult);
    free(intArrayA);
    free(intArrayB);
    free(intArrayResult);
    return 0;
}

