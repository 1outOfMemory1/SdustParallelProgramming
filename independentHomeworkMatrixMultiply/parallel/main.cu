#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#define ND 2000
#define size 25
using namespace std;


int a[ND][ND];
int b[ND][ND];
int c[ND][ND];


__global__ void MatMul(int *M,int *N,int *P,int width)
{

    int Col = blockIdx.x*blockDim.x + threadIdx.x; // cloumn
    int Row = blockIdx.y*blockDim.y + threadIdx.y; // row

    float elem1 = 0.0,elem2 = 0.0,value = 0.0;
    for(int i = 0;i < width;i++)
    {
        elem1 = M[Col * width + i];//取M矩阵的一行
        elem2 = N[i * width + Row];//取N矩阵的一列
        value += elem1 * elem2;//求和
    }

    P[Col * width + Row] = value;
}


int main()
{

//    int **a=new int*[ND],**b=new int*[ND],**c=new int*[ND];
//    for(int i=0;i<ND;i++){
//        a[i] = new int[ND];
//        b[i] = new int[ND];
//        c[i] = new int[ND];
//    }

    //初始化
    int mm;
    for(int mm = 0;mm < ND;mm++)
    {
        for(int j = 0;j < ND;j++)
        {
            a[mm][j] = 1;
            b[mm][j] = 2;
            c[mm][j] = 0;
        }
    }
//
//    for(int i=0;i<ND;i++){
//        for(int j=0;j<ND;j++){
//            printf("%d ",a[i][j]);
//        }
//    }
//
//    for(int i=0;i<ND;i++){
//        for(int j=0;j<ND;j++){
//            printf("%d ",b[i][j]);
//        }
//    }
//    for(int i=0;i<ND;i++){
//        for(int j=0;j<ND;j++){
//            printf("%d ",c[i][j]);
//        }
//    }




    int *M,*N,*P;

    int width = ND;
    dim3 gridSize(ND/size,ND/size);
    dim3 blockSize(size,size);

    hipEvent_t start,stop;
    float elapsedTime = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //设备端内存分配
    hipMalloc((void**)&M,ND * ND * sizeof(int));
    hipMalloc((void**)&N,ND * ND * sizeof(int));
    hipMalloc((void**)&P,ND * ND * sizeof(int));



    int Size = ND * ND;
    //数据拷贝，主机到设备
    hipMemcpy(M,a,Size * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(N,b,Size * sizeof(int),hipMemcpyHostToDevice);

    hipEventRecord(start,0);
    MatMul<<<gridSize,blockSize>>>(M,N,P,width);//调用核函数
    hipDeviceSynchronize();
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);


    hipError_t error =  hipMemcpy(c,P,Size * sizeof(int),hipMemcpyDeviceToHost);

    cout<< error;



    for(int i=0;i<ND;i++){
        for(int j=0;j<ND;j++){
            printf("%d i:%d j:%d ",c[i][j],i,j);
        }
    }


    //释放设备内存
    hipFree(M);
    hipFree(N);
    hipFree(P);

    return 0;
}
