
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
const int n = 7;
//共有x个块，对应矩阵的x行
//每块内有y个线程，对应矩阵的y列
__global__ void kernel(int * d_a,int * d_b,int * d_result)
{
    //两维的d_result[][]矩阵存放结果，blockIdx.x代表线程所处理的行坐标，
    //threadIdx.x代表线程所处理的列坐标
    d_result[blockIdx.x*n+threadIdx.x]=0;
    for(int i=0;i<n;i++)
        d_result[blockIdx.x*n+threadIdx.x]+=d_a[blockIdx.x*n+i]*d_b[i*n+threadIdx.x];
}
int main()
{
    //指向CPU端内存的指针

    int  h_a[n*n],h_b[n*n],h_result[n*n];

    //为两个矩阵赋初值
    for(int i=0;i<n;i++)
        for(int j=0;j<n;j++)
            h_a[i*n+j] = h_b[i*n+j] = (10);

    //指向GPU端内存的指针
    int * d_a , *d_b , *d_result ;

    //为GPU中的数据分配内存
    hipMalloc( (void**)&d_a,sizeof(int)*n*n  );
    hipMalloc( (void**)&d_b,sizeof(int)*n*n  );
    hipMalloc( (void**)&d_result,sizeof(int)*n*n  );

    //拷贝CPU中的数据到GPU
    hipMemcpy(d_a,h_a,sizeof(int)*n*n,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,sizeof(int)*n*n,hipMemcpyHostToDevice);

    //调用内核函数，启动n个block，每个block里有n个线程
    kernel<<<n,n>>>(d_a,d_b,d_result);

    //把GPU中算出来的数据拷回CPU
    hipMemcpy(h_result,d_result,sizeof(int)*n*n,hipMemcpyDeviceToHost);

    //显示
    for(int i=0;i<n;i++)
        for(int j=0;j<n;j++)
        {
            cout<<h_result[i*n+j] <<"  ";
            if(j==n-1)
                cout<<'\n';
//            else
//                cout<<'\t'<<'\t';
        }

}
